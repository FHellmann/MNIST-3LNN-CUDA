#include "NeuralNetworkCUDA.h"

#include <iostream>

using namespace std;

__host__ NeuralNetworkCUDA::NeuralNetworkCUDA(const int inpCount, const int hidCount,
		const int outCount, const double learningRate) :
			NeuralNetwork(learningRate) {
}

__host__ NeuralNetworkCUDA::~NeuralNetworkCUDA() {
}

__host__ void NeuralNetworkCUDA::train(MNISTImageDataset const& images,
		MNISTLableDataset const& labels, double const training_error_threshold,
		double const max_derivation) {

	if (images.size() <= 0)
		return;
	if (labels.size() <= 0)
		return;

	// Collect memory in RAM
	uint8_t* imgData = new uint8_t[images.front().total() * images.size()];
	uint8_t* it = imgData;
	for (cv::Mat const& img : images) {
		if (img.isContinuous()) {
			std::copy(img.datastart, img.dataend, it);
		} else {
			cerr << "cv::Mat is not continuous." << endl;
		}
		it += img.total() * img.elemSize();
	}
//
//	hipError_t err;

	// Allocate cuda memory
//	err = hipMalloc((void**) &d_A, A.size() * sizeof(float));
//	assert(err == hipSuccess);

	// Copy data to graphics card
//	err = hipMemcpy(d_A, A.data(), A.size() * sizeof(float),
//			hipMemcpyHostToDevice);
//	assert(err == hipSuccess);
	delete[] imgData;
	imgData = nullptr;

	// Configure Grid, i.e. setup Blocks and Threads
//	dim3 threadsPerBlock(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
//	dim3 numBlocks(C.cols() / MATRIX_SIZE_DIVISOR,
//			C.rows() / MATRIX_SIZE_DIVISOR);
//	cout << "Threads per block: (" << threadsPerBlock.x << ", "
//			<< threadsPerBlock.y << ")" << endl;
//	cout << "Blocks:            (" << numBlocks.x << ", " << numBlocks.y << ")"
//			<< endl;

	// Call graphics card functions
//	d_mul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A.rows());

	// Retreive the data
//	err = hipMemcpy(C.data(), d_C, C.size() * sizeof(float),
//			hipMemcpyDeviceToHost);
//	assert(err == hipSuccess);

	// Copy it back to neural network datastructure

	// Free the cuda buffers
//	hipFree(d_A);
//	d_A = nullptr;
}
