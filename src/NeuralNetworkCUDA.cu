#include "hip/hip_runtime.h"
#include "NeuralNetworkCUDA.h"

#include <iostream>
#include <cmath>

using namespace std;

__host__ NeuralNetworkCUDA::NeuralNetworkCUDA(const int inpCount,
		const int hidCount, const int outCount, const double learningRate) :
		NeuralNetwork(inpCount, hidCount, outCount, learningRate) {
}

__host__ NeuralNetworkCUDA::~NeuralNetworkCUDA() {
}

#define MATRIX_SIZE_DIVISOR 28
#define NUM_DIGITS 10

struct Matrix {
	enum Layout {
		ROW_MAJOR,
		COLUMN_MAJOR
	};

	size_t rows;
	size_t cols;
	Layout layout = ROW_MAJOR;
	float* data;
};

struct GPUTrainingParameters {
	/* Training data. */
	float* images;
	float* labels;

	/* Training data parameters. */
	size_t numExamples;
	size_t numHiddenNodes;
	size_t width;
	size_t height;
	// Number of images per training
	size_t batchSize;

	/* Weight matrices. */
	Matrix W12;
	Matrix W23;

	/* Biases */
	Matrix bias2;
	Matrix bias3;

	/* Layer data */
	Matrix output2;
	Matrix output3;

	NeuralNetwork::ActFctType activationFunction2;
	NeuralNetwork::ActFctType activationFunction3;

	/* Training parameters. */
	float errorThreshold;
	float maxDerivation;

	/* Temporary buffers, e.g. for back propagation. */
	Matrix tmp3;
	Matrix tmp2;
};

struct GPUSharedMemoryLayout {
	size_t W1_pos = 0;
	size_t W1_size = 0;
	size_t W2_pos = 0;
	size_t W2_size = 0;
	size_t inputBias_pos = 0;
	size_t inputBias_size = 0;
	size_t hiddenOutput_pos = 0;
	size_t hiddenOutput_size = 0;
	size_t hiddenBias_pos = 0;
	size_t hiddenBias_size = 0;
	size_t outputOutput_pos = 0;
	size_t outputOutput_size = 0;
	size_t outputBias_pos = 0;
	size_t outputBias_size = 0;
	size_t image_pos = 0;
	size_t image_size = 0;
} gpuSharedMemoryLayout;

__device__ float* d_matrix_pget(Matrix const& M, size_t const y, size_t const x) {
	if (M.layout == Matrix::ROW_MAJOR) {
		return M.data + (x + y * M.cols);
	} else {
		return M.data + (x * M.rows + y);
	}
}

__device__ float d_matrix_get(Matrix const& M, size_t const y, size_t const x) {
	return *d_matrix_pget(M, y, x);
}

__device__ void d_matrix_set(Matrix const& M, size_t const y, size_t const x, float const value) {
	if (M.layout == Matrix::ROW_MAJOR) {
		M.data[x + y * M.cols] = value;
	} else {
		M.data[x * M.rows + y] = value;
	}
}

__device__ size_t d_matrix_size(Matrix const& A) {
	return A.rows * A.cols;
}

size_t matrix_size(Matrix const& A) {
	return A.rows * A.cols;
}

__device__ Matrix d_matrix_transpose(Matrix const& A) {
	Matrix T;
	T.rows = A.cols;
	T.cols = A.rows;
	T.layout = Matrix::ROW_MAJOR;
	if (A.layout == Matrix::ROW_MAJOR) {
		T.layout = Matrix::COLUMN_MAJOR;
	}
	return T;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void d_feed_forward(GPUTrainingParameters const);
__global__ void d_back_propagate(GPUTrainingParameters const);

__host__ void NeuralNetworkCUDA::train(MNISTImageDataset const& images,
		MNISTLableDataset const& labels, double const training_error_threshold,
		double const max_derivation) {

	if (images.size() <= 0)
		return;
	if (labels.size() <= 0)
		return;

	Layer* const inputLayer  = getLayer(INPUT);
	Layer* const hiddenLayer = getLayer(HIDDEN);
	Layer* const outputLayer = getLayer(OUTPUT);

	GPUTrainingParameters trainingParams;
	trainingParams.numExamples = images.size();
	trainingParams.width = images.front().cols;
	trainingParams.height = images.front().rows;
	trainingParams.numHiddenNodes = hiddenLayer->nodes.size();
	trainingParams.errorThreshold = training_error_threshold;
	trainingParams.maxDerivation = max_derivation;
	trainingParams.batchSize = MATRIX_SIZE_DIVISOR;

	hipError_t err;

	//
	// Allocate cuda memory
	//
	size_t const singleImgPixCount = images.front().total();
	size_t const allImgBufElements = singleImgPixCount * images.size();

	// Images
	err = hipMalloc((void**) &trainingParams.images, allImgBufElements * sizeof(float));
	assert(err == hipSuccess);

	// Labels
	err = hipMalloc((void**) &trainingParams.labels, labels.size() * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the first weight matrix
	trainingParams.W12.rows = hiddenLayer->nodes.size();
	trainingParams.W12.cols = inputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W12.data, matrix_size(trainingParams.W12) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the hidden layer bias vector
	trainingParams.bias2.rows = hiddenLayer->nodes.size();
	trainingParams.bias2.cols = 1;
	err = hipMalloc((void**) &trainingParams.bias2.data, matrix_size(trainingParams.bias2) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the second weight matrix
	trainingParams.W23.rows = outputLayer->nodes.size();
	trainingParams.W23.cols = hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W23.data, matrix_size(trainingParams.W23) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer bias vector
	trainingParams.bias3.rows = outputLayer->nodes.size();
	trainingParams.bias3.cols = 1;
	err = hipMalloc((void**) &trainingParams.bias3.data, matrix_size(trainingParams.bias3) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer output vectors
	trainingParams.output2.rows = trainingParams.numHiddenNodes;
	trainingParams.output2.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.output2.data, matrix_size(trainingParams.output2) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer output vectors
	trainingParams.output3.rows = outputLayer->nodes.size();
	trainingParams.output3.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.output3.data, matrix_size(trainingParams.output3) * sizeof(float));
	assert(err == hipSuccess);

	// Temporary storage of the size of the output layer output vectors
	trainingParams.tmp3.rows = outputLayer->nodes.size();
	trainingParams.tmp3.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.tmp3.data, matrix_size(trainingParams.tmp3) * sizeof(float));
	assert(err == hipSuccess);

	// Temporary storage of the size of the hidden layer output vectors
	trainingParams.tmp2.rows = hiddenLayer->nodes.size();
	trainingParams.tmp2.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.tmp2.data, matrix_size(trainingParams.tmp2) * sizeof(float));
	assert(err == hipSuccess);

	//
	// Collect memory in RAM
	//
	float* imgData = new float[allImgBufElements];
	float* dst = imgData;
	for (cv::Mat const& img : images) {
		for (uint8_t* src = img.datastart; src != img.dataend;) {
			*(dst++) = static_cast<float>(*(src++));
		}
	}

	float* flabels = new float[labels.size()];
	dst = flabels;
	for (uint8_t const& l : labels) {
		*(dst++) = static_cast<float>(l);
	}

	float* W12 = new float[matrix_size(trainingParams.W12)];
	float* W23 = new float[matrix_size(trainingParams.W23)];
	float* bias2 = new float[matrix_size(trainingParams.bias2)];
	float* bias3 = new float[matrix_size(trainingParams.bias3)];

	//
	// Collect the initial weights and biases in buffers for submission to the GPU.
	//
	trainingParams.activationFunction2 = hiddenLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < hiddenLayer->nodes.size(); ++j) {
			Layer::Node* node = hiddenLayer->nodes[j];
			bias2[j] = node->bias;
			for (size_t i = 0; i < node->weights.size(); ++i) {
				W12[k] = node->weights[i];
				++k;
			}
		}
	}

	trainingParams.activationFunction3 = outputLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < outputLayer->nodes.size(); ++j) {
			Layer::Node* node = outputLayer->nodes[j];
			bias3[j] = node->bias;
			for (size_t i = 0; i < node->weights.size(); ++i) {
				W23[k] = node->weights[i];
				++k;
			}
		}
	}

	//
	// Copy data to graphics card
	//
	err = hipMemcpy(trainingParams.images, imgData, allImgBufElements * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.labels, flabels, labels.size() * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.W12.data, W12, matrix_size(trainingParams.W12) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.bias2.data, bias2, matrix_size(trainingParams.bias2) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.W23.data, W23, matrix_size(trainingParams.W23) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.bias3.data, bias3, matrix_size(trainingParams.bias3) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);

	delete[] imgData;
	imgData = nullptr;
	delete[] flabels;
	flabels = nullptr;

//	size_t sharedMemorySize = 0;

	// Size of the first weight matrix
//	gpuSharedMemoryLayout.W1_pos = 0;
//	gpuSharedMemoryLayout.W1_size = trainingParams.W1_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W1_size;
//
//	// Size of the second weight matrix
//	gpuSharedMemoryLayout.W2_pos = gpuSharedMemoryLayout.W1_pos + gpuSharedMemoryLayout.W1_size;
//	gpuSharedMemoryLayout.W2_size = trainingParams.W2_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W2_size;
//
//	// Size of the hidden layer output nodes
//	gpuSharedMemoryLayout.hiddenOutput_pos = gpuSharedMemoryLayout.W2_pos + gpuSharedMemoryLayout.W2_size;
//	gpuSharedMemoryLayout.hiddenOutput_size = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenOutput_size;
//
//	// Size of the output layer output values
//	gpuSharedMemoryLayout.outputOutput_pos = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.outputOutput_size = outputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.outputOutput_size;
//
//	// Size of the hidden bias vector
//	gpuSharedMemoryLayout.hiddenBias_pos   = gpuSharedMemoryLayout.outputOutput_pos + gpuSharedMemoryLayout.outputOutput_size;
//	gpuSharedMemoryLayout.hiddenBias_size  = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenBias_size;
//
//	// Size of the input bias vector
//	gpuSharedMemoryLayout.inputBias_pos    = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.inputBias_size   = inputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.inputBias_size;
//
//	// Size of the input vector
//	gpuSharedMemoryLayout.image_pos        = gpuSharedMemoryLayout.inputBias_pos + gpuSharedMemoryLayout.inputBias_size;
//	gpuSharedMemoryLayout.image_size       = inputLayer->nodes.size() * sizeof(uint8_t);
//	sharedMemorySize += gpuSharedMemoryLayout.image_size;

	// Configure Grid, i.e. setup Blocks and Threads
	dim3 numBlocks(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
	dim3 threadsPerBlock(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
	cout << "Blocks:            (" << numBlocks.x << ", " << numBlocks.y << ")"
			<< endl;
	cout << "Threads per block: (" << threadsPerBlock.x << ", "
			<< threadsPerBlock.y << ")" << endl;

	// Call graphics card functions
	d_feed_forward<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	d_back_propagate<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	//
	// Retreive the data
	//

	// Copy it back to neural network data structure
	err = hipMemcpy(W12, trainingParams.W12.data, matrix_size(trainingParams.W12) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(W23, trainingParams.W23.data, matrix_size(trainingParams.W23) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias2, trainingParams.bias2.data, matrix_size(trainingParams.bias2) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias3, trainingParams.bias3.data, matrix_size(trainingParams.bias3) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	// Free the cuda buffers
	hipFree (trainingParams.images);
	trainingParams.images = nullptr;
	hipFree (trainingParams.labels);
	trainingParams.labels = nullptr;
	hipFree (trainingParams.W12.data);
	trainingParams.W12.data = nullptr;
	hipFree (trainingParams.W23.data);
	trainingParams.W23.data = nullptr;
	hipFree (trainingParams.bias2.data);
	trainingParams.bias2.data = nullptr;
	hipFree (trainingParams.bias3.data);
	trainingParams.bias3.data = nullptr;
	hipFree (trainingParams.output2.data);
	trainingParams.output2.data = nullptr;
	hipFree (trainingParams.output3.data);
	trainingParams.output3.data = nullptr;
	hipFree (trainingParams.tmp3.data);
	trainingParams.tmp3.data = nullptr;
	hipFree (trainingParams.tmp2.data);
	trainingParams.tmp2.data = nullptr;

	//
	// Copy the weight data into the c++ data structure.
	//
	trainingParams.activationFunction2 = hiddenLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < hiddenLayer->nodes.size(); ++j) {
			Layer::Node* node = hiddenLayer->nodes[j];
			node->bias = bias2[j];
			for (size_t i = 0; i < node->weights.size(); ++i) {
				node->weights[i] = W12[k];
				++k;
			}
		}
	}

	trainingParams.activationFunction3 = outputLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < outputLayer->nodes.size(); ++j) {
			Layer::Node* node = outputLayer->nodes[j];
			node->bias = bias3[j];
			for (size_t i = 0; i < node->weights.size(); ++i) {
				node->weights[i] = W23[k];
				++k;
			}
		}
	}

	// Delete the host buffers
	delete[] W12;
	W12 = nullptr;
	delete[] W23;
	W23 = nullptr;
	delete[] bias2;
	bias2 = nullptr;
	delete[] bias3;
	bias3 = nullptr;
}

__device__ void d_print(GPUTrainingParameters const params) {
	printf("TrainingParams:\n"
			"  W12: %p\n"
		    "  W1_len: %lu\n"
			"  W2: %p\n"
			"  W2_len: %lu\n"
			"  errorThreshold: %f\n"
			"  width: %lu\n"
			"  height: %lu\n"
			"  numExamples: %lu\n"
			"  numHiddenNodes: %lu\n",
			params.W12.data,
			d_matrix_size(params.W12),
			params.W23.data,
			d_matrix_size(params.W23),
			params.errorThreshold,
			params.width,
			params.height,
			params.numExamples,
			params.numHiddenNodes);
}

/* Matrix manipulation operations. */
__device__ void d_mul_base(Matrix const& C, Matrix const& A, Matrix const& B, void(*op)(float*, float const, float const));
__device__ void d_mul(Matrix const& C, Matrix const& A, Matrix const& B);
__device__ void d_mul_add(Matrix const& C, Matrix const& A, Matrix const& B);
__device__ void d_cwise_op(Matrix const& C, Matrix const& A, Matrix const& B, void(*op)(float*, float const, float const));
__device__ void d_cwise_mul(Matrix const& C, Matrix const& A, Matrix const& B);
__device__ void d_cwise_sub(Matrix const& C, Matrix const& A, Matrix const& B);

/* Neural network operations. */
__device__ void d_apply_activation(Matrix const&, NeuralNetwork::ActFctType);
__device__ void d_apply_activation_derivative(Matrix const&, NeuralNetwork::ActFctType);
__device__ void d_back_propagate_output(GPUTrainingParameters const);
__device__ void d_back_propagate_hidden(GPUTrainingParameters const);
__device__ void d_fill_target_output(GPUTrainingParameters const, Matrix const&);
__device__ void d_set_bias(Matrix const& output, Matrix const& bias);
__device__ void d_fill_random(Matrix const&);

__global__ void d_feed_forward(GPUTrainingParameters const params) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_feed_forward\n");
	}

	Matrix imgs;
	imgs.rows = params.width * params.height;
	imgs.cols = params.batchSize;
	imgs.layout = Matrix::COLUMN_MAJOR;
	imgs.data = params.images; // Global data pointer, column major, yields one image in each column vector.

	d_set_bias(params.output2, params.bias2);
	d_mul_add(params.output2, params.W12, imgs);
	d_apply_activation(params.output2, params.activationFunction2);

	d_set_bias(params.output3, params.bias3);
	d_mul_add(params.output3, params.W23, params.output2);
	d_apply_activation(params.output3, params.activationFunction3);
}

__global__ void d_back_propagate(GPUTrainingParameters const params) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_back_propagate\n");
	}

	d_back_propagate_output(params);
	d_back_propagate_hidden(params);
}

__device__ void d_back_propagate_output(GPUTrainingParameters const params) {

	Matrix targetOutput = params.tmp3;

	// Compute the target output based on the labels
	d_fill_target_output(params, targetOutput);

	// Save the difference into the target output buffer
	Matrix difference = targetOutput;
	// Reuse the output buffer for saving the error, for now. Perhaps this is a problem later on.
	Matrix error = params.output3;

	d_cwise_sub(difference, targetOutput, params.output3);
	d_apply_activation_derivative(params.output3, params.activationFunction3);
	d_cwise_mul(error, params.output3, difference);

	// Important to make a local copy.
	// Otherwise every thread would transpose the matrix which
	// would lead to undefined behavior.
	Matrix output2 = d_matrix_transpose(params.output2);
	d_mul_add(params.W23, error, output2);
}

__device__ void d_back_propagate_hidden(GPUTrainingParameters const params) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_back_propagate_hidden\n");
	}

	// The weight updates are computed by
	// W23^T * e3 * ∇σ * input^T

	// Important to make a local copy.
	// Otherwise every thread would transpose the matrix which
	// would lead to undefined behavior.
	Matrix W23 = d_matrix_transpose(params.W23);

	// See d_back_propagation_output
	Matrix error = params.output3;

	Matrix images;
	images.rows = params.width * params.height;
	images.cols = params.batchSize;
	images.layout = Matrix::COLUMN_MAJOR;
	images.data = params.images;
	images = d_matrix_transpose(images);

	d_apply_activation_derivative(params.output2, params.activationFunction2);
	d_mul(params.tmp2, W23, error);
	d_cwise_mul(params.tmp2, params.output2, params.tmp2);
//	d_mul_add(params.W12, params.tmp2, images);
}

__device__ void d_apply_activation(Matrix const& A, NeuralNetwork::ActFctType functionType) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_activate_layer\n");
	}

	// Target index for this thread.
	size_t const idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y + blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	// If the target index would handle an element outside of the data buffer, terminate.
	if (idx >= A.cols * A.rows) {
		return;
	}

	switch (functionType) {
	case NeuralNetwork::SIGMOID:
		A.data[idx] = 1.0f / (1.0f + exp(-A.data[idx]));
		break;
	case NeuralNetwork::TANH:
		A.data[idx] = tanh(A.data[idx]);
		break;
	}
}

__device__ void d_apply_activation_derivative(Matrix const& A, NeuralNetwork::ActFctType functionType) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_apply_activation_derivative\n");
	}

	// Target index for this thread.
	size_t const idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y + blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	// If the target index would handle an element outside of the data buffer, terminate.
	if (idx >= A.rows * A.cols) {
		return;
	}

	switch (functionType) {
	case NeuralNetwork::SIGMOID:
		A.data[idx] = A.data[idx] * (1.0f - A.data[idx]);
		break;
	case NeuralNetwork::TANH:
		float t = tanh(A.data[idx]);
		A.data[idx] = 1.0f - t * t;
		break;
	}
	//printf("actFctDeriv(%lu) = %f\n", idx, data[idx]);
}

__device__ void d_fill_target_output(GPUTrainingParameters const params, Matrix const& targetOutput) {

	if (targetOutput.rows != NUM_DIGITS) {
		printf("d_fill_target_output: wrong number of rows. Given %lu, expected %u\n", targetOutput.rows, NUM_DIGITS);
		return;
	}

	size_t srcIdx = threadIdx.x + blockIdx.x * blockDim.x;
	size_t targetX = threadIdx.x + blockIdx.x * blockDim.x;
	size_t targetY = threadIdx.y + blockIdx.y * blockDim.y;

	if (targetX >= targetOutput.cols || targetY >= targetOutput.rows) {
		return;
	}

	float const v = (threadIdx.y == params.labels[srcIdx]) ? 1.0f : 0.0f;
	d_matrix_set(targetOutput, targetY, targetX, v);
}

__device__ void d_set_bias(Matrix const& output, Matrix const& bias) {

	if (bias.rows != output.rows) {
		printf("d_set_bias: Bias and output dimensions mismatch. Expected same height but bias was %lu and output was %lu\n", bias.rows, output.rows);
		return;
	}

	if (bias.cols > 1) {
		printf("d_set_bias: Bias column dimension is %lu > 1. Not handled.\n", bias.cols);
		return;
	}

	size_t const targetX = threadIdx.x + blockIdx.x * blockDim.x;
	size_t const targetY = threadIdx.y + blockIdx.y * blockDim.y;

	if (targetX >= output.cols || targetY >= output.rows) {
		return;
	}

	d_matrix_set(output, targetY, targetX, d_matrix_get(bias, targetY, 1));
}

__device__ void d_assign(float* c, float const a, float const b) {
	*c = b;
}

__device__ void d_add(float* c, float const a, float const b) {
	*c = a + b;
	//printf("d_add(%f, %f, %f\n)", *a, b, c);
}

__device__ void d_sub(float* c, float const a, float const b) {
	*c = a - b;
	//printf("d_add(%f, %f, %f)\n", *c, a, b);
}

__device__ void d_mul(float* c, float const a, float const b) {
	*c = a * b;
}

__device__ void d_mul(Matrix const& C, Matrix const& A, Matrix const& B) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_mul\n");
	}
	d_mul_base(C, A, B, &d_assign);
}

__device__ void d_mul_add(Matrix const& C, Matrix const& A, Matrix const& B) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_mul_add\n");
	}
	d_mul_base(C, A, B, &d_add);
}

/**
 * Computes C = AB where the dimensions of A and be have to be a multiple of MATRIX_SIZE_DIVISOR.
 *
 * @param[in] A first factor of the matrix multiplication.
 * @param[in] B second factor of the multiplication.
 * @param[out] C Matrix holding the result. Must provide enough storage space.
 */
__device__ void d_mul_base(Matrix const& C, Matrix const& A, Matrix const& B, void(*op)(float*, float const, float const)) {

	if (A.cols != B.rows) {

		printf("d_mul_base: Incompatible matrices: (%lu, %lu) x (%lu, %lu)\n", A.rows, A.cols, B.rows, B.cols);
		return;
	}

	// The block caches are row major.
	__shared__ float blockCacheA[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];
	__shared__ float blockCacheB[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];

	// If this thread has nothing to do, because it would access invalid memory, exit
	if (blockIdx.x * MATRIX_SIZE_DIVISOR + threadIdx.x > C.cols ||
		blockIdx.y * MATRIX_SIZE_DIVISOR + threadIdx.y > C.rows) {
		return;
	}

	float threadValue = 0.0f;
	unsigned int const numSubBlocks = A.cols / MATRIX_SIZE_DIVISOR;
	for (int k = 0; k < numSubBlocks; ++k)
	{
		size_t const xA = k * MATRIX_SIZE_DIVISOR + threadIdx.x;
		size_t const yA = blockIdx.y * MATRIX_SIZE_DIVISOR + threadIdx.y;
		blockCacheA[threadIdx.y][threadIdx.x] = d_matrix_get(A, yA, xA);

		size_t const xB = blockIdx.x * MATRIX_SIZE_DIVISOR + threadIdx.x;
		size_t const yB = k * MATRIX_SIZE_DIVISOR + threadIdx.y;
		blockCacheB[threadIdx.y][threadIdx.x] = d_matrix_get(B, yB, xB);

		__syncthreads();

		#pragma unroll
		for (int i = 0; i < MATRIX_SIZE_DIVISOR; ++i)
		{
			threadValue += blockCacheA[threadIdx.y][i] * blockCacheB[i][threadIdx.x];
		}

		__syncthreads();
	}

	size_t const x = blockIdx.x * MATRIX_SIZE_DIVISOR + threadIdx.x;
	size_t const y = blockIdx.y * MATRIX_SIZE_DIVISOR + threadIdx.y;
	float* const pValue = d_matrix_pget(C, y, x);
	op(pValue, *pValue, threadValue);
}

__device__ void d_cwise_sub(Matrix const& C, Matrix const& A, Matrix const& B) {
	d_cwise_op(C, A, B, &d_sub);
}

__device__ void d_cwise_mul(Matrix const& C, Matrix const& A, Matrix const& B) {
	d_cwise_op(C, A, B, &d_mul);
}

__device__ void d_cwise_op(Matrix const& C, Matrix const& A, Matrix const& B, void(*op)(float*, float const, float const)) {

	if (A.cols != B.cols || A.rows != B.rows || B.cols != C.cols || B.rows != C.rows) {

		printf("d_cwise_op: Incompatible matrices: (%lu, %lu) + (%lu, %lu) = (%lu, %lu)\n", A.rows, A.cols, B.rows, B.cols, C.rows, C.cols);
		return;
	}

	size_t const x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t const y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= A.cols || y >= A.rows) {
		return;
	}

	//C.data[idxC] = A.data[idxA] - B.data[idxB];
	op(d_matrix_pget(C, y, x), d_matrix_get(A, y, x), d_matrix_get(B, y, x));
}

__device__ void d_fill_random(Matrix const& A) {

	size_t const targetX = threadIdx.x + blockIdx.x * blockDim.x;
	size_t const targetY = threadIdx.y + blockIdx.y * blockDim.y;

	if (targetX >= A.cols || targetY >= A.rows) {
		return;
	}

	d_matrix_set(A, targetY, targetX, static_cast<float>(targetX));
}
