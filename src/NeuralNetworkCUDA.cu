#include "hip/hip_runtime.h"
#include "NeuralNetworkCUDA.h"

#include <iostream>
#include <cmath>

using namespace std;

__host__ NeuralNetworkCUDA::NeuralNetworkCUDA(const int inpCount,
		const int hidCount, const int outCount, const double learningRate) :
		NeuralNetwork(inpCount, hidCount, outCount, learningRate) {
}

__host__ NeuralNetworkCUDA::~NeuralNetworkCUDA() {
}

#define MATRIX_SIZE_DIVISOR 28
#define NUM_DIGITS 10

struct GPUTrainingParameters {
	/* Training data. */
	float* images;
	float* labels;

	/* Training data parameters. */
	size_t numExamples;
	size_t numHiddenNodes;
	size_t width;
	size_t height;
	// Number of images per training
	size_t batchSize;

	/* Weight matrices. */
	float* W12;
	size_t W12_len;
	float* W23;
	size_t W23_len;

	/* Biases */
	float* bias2;
	size_t bias2_len;
	float* bias3;
	size_t bias3_len;

	/* Layer data */
	float* output2;
	size_t output2_len;
	float* output3;
	size_t output3_len;

	NeuralNetwork::ActFctType activationFunction2;
	NeuralNetwork::ActFctType activationFunction3;

	/* Training parameters. */
	float errorThreshold;
	float maxDerivation;
};

struct GPUSharedMemoryLayout {
	size_t W1_pos = 0;
	size_t W1_size = 0;
	size_t W2_pos = 0;
	size_t W2_size = 0;
	size_t inputBias_pos = 0;
	size_t inputBias_size = 0;
	size_t hiddenOutput_pos = 0;
	size_t hiddenOutput_size = 0;
	size_t hiddenBias_pos = 0;
	size_t hiddenBias_size = 0;
	size_t outputOutput_pos = 0;
	size_t outputOutput_size = 0;
	size_t outputBias_pos = 0;
	size_t outputBias_size = 0;
	size_t image_pos = 0;
	size_t image_size = 0;
} gpuSharedMemoryLayout;

struct Matrix {
	enum Layout {
		ROW_MAJOR,
		COLUMN_MAJOR
	};

	size_t rows;
	size_t cols;
	Layout layout = ROW_MAJOR;
	float* data;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void d_feed_forward(GPUTrainingParameters const);
__global__ void d_back_propagate(GPUTrainingParameters const);

__host__ void NeuralNetworkCUDA::train(MNISTImageDataset const& images,
		MNISTLableDataset const& labels, double const training_error_threshold,
		double const max_derivation) {

	if (images.size() <= 0)
		return;
	if (labels.size() <= 0)
		return;

	// Collect memory in RAM
	size_t const singleImgPixCount = images.front().total();
	size_t const allImgBufElements = singleImgPixCount * images.size();
	float* imgData = new float[allImgBufElements];
	float* dst = imgData;
	for (cv::Mat const& img : images) {
		for (uint8_t* src = img.datastart; src != img.dataend;) {
			*(dst++) = static_cast<float>(*(src++));
		}
	}

	float* flabels = new float[labels.size()];
	dst = flabels;
	for (uint8_t const& l : labels) {
		*(dst++) = static_cast<float>(l);
	}

	hipError_t err;

	Layer* inputLayer  = getLayer(INPUT);
	Layer* hiddenLayer = getLayer(HIDDEN);
	Layer* outputLayer = getLayer(OUTPUT);

	GPUTrainingParameters trainingParams;
	trainingParams.numExamples = images.size();
	trainingParams.width = images.front().cols;
	trainingParams.height = images.front().rows;
	trainingParams.numHiddenNodes = hiddenLayer->nodes.size();
	trainingParams.errorThreshold = training_error_threshold;
	trainingParams.maxDerivation = max_derivation;
	trainingParams.batchSize = MATRIX_SIZE_DIVISOR;

	//
	// Allocate cuda memory
	//

	// Images
	err = hipMalloc((void**) &trainingParams.images, allImgBufElements * sizeof(float));
	assert(err == hipSuccess);

	// Labels
	err = hipMalloc((void**) &trainingParams.labels, labels.size() * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the first weight matrix
	trainingParams.W12_len = inputLayer->nodes.size() * hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W12, trainingParams.W12_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the hidden layer bias vector
	trainingParams.bias2_len = hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.bias2, trainingParams.bias2_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the second weight matrix
	trainingParams.W23_len = hiddenLayer->nodes.size() * outputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W23, trainingParams.W23_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer bias vector
	trainingParams.bias3_len = outputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.bias3, trainingParams.bias3_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer output vectors
	trainingParams.output2_len = hiddenLayer->nodes.size() * trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.output2, trainingParams.output2_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer output vectors
	trainingParams.output3_len = outputLayer->nodes.size() * trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.output3, trainingParams.output3_len * sizeof(float));
	assert(err == hipSuccess);

	//
	// Copy data to graphics card
	//
	err = hipMemcpy(trainingParams.images, imgData, allImgBufElements * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.labels, flabels, labels.size() * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);

	delete[] imgData;
	imgData = nullptr;
	delete[] flabels;
	flabels = nullptr;

//	size_t sharedMemorySize = 0;

	// Size of the first weight matrix
//	gpuSharedMemoryLayout.W1_pos = 0;
//	gpuSharedMemoryLayout.W1_size = trainingParams.W1_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W1_size;
//
//	// Size of the second weight matrix
//	gpuSharedMemoryLayout.W2_pos = gpuSharedMemoryLayout.W1_pos + gpuSharedMemoryLayout.W1_size;
//	gpuSharedMemoryLayout.W2_size = trainingParams.W2_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W2_size;
//
//	// Size of the hidden layer output nodes
//	gpuSharedMemoryLayout.hiddenOutput_pos = gpuSharedMemoryLayout.W2_pos + gpuSharedMemoryLayout.W2_size;
//	gpuSharedMemoryLayout.hiddenOutput_size = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenOutput_size;
//
//	// Size of the output layer output values
//	gpuSharedMemoryLayout.outputOutput_pos = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.outputOutput_size = outputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.outputOutput_size;
//
//	// Size of the hidden bias vector
//	gpuSharedMemoryLayout.hiddenBias_pos   = gpuSharedMemoryLayout.outputOutput_pos + gpuSharedMemoryLayout.outputOutput_size;
//	gpuSharedMemoryLayout.hiddenBias_size  = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenBias_size;
//
//	// Size of the input bias vector
//	gpuSharedMemoryLayout.inputBias_pos    = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.inputBias_size   = inputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.inputBias_size;
//
//	// Size of the input vector
//	gpuSharedMemoryLayout.image_pos        = gpuSharedMemoryLayout.inputBias_pos + gpuSharedMemoryLayout.inputBias_size;
//	gpuSharedMemoryLayout.image_size       = inputLayer->nodes.size() * sizeof(uint8_t);
//	sharedMemorySize += gpuSharedMemoryLayout.image_size;

	hipMemset(trainingParams.W12, 0.0, trainingParams.W12_len * sizeof(float));
	hipMemset(trainingParams.W23, 0.0, trainingParams.W23_len * sizeof(float));
	hipMemset(trainingParams.bias2, 0.0, trainingParams.bias2_len * sizeof(float));
	hipMemset(trainingParams.bias3, 0.0, trainingParams.bias3_len * sizeof(float));

	// Configure Grid, i.e. setup Blocks and Threads
	dim3 numBlocks(2,2);
	dim3 threadsPerBlock(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
	cout << "Blocks:            (" << numBlocks.x << ", " << numBlocks.y << ")"
			<< endl;
	cout << "Threads per block: (" << threadsPerBlock.x << ", "
			<< threadsPerBlock.y << ")" << endl;

	// Call graphics card functions
	d_feed_forward<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	d_back_propagate<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	//
	// Retreive the data
	//
	float* W12 = new float[trainingParams.W12_len];
	float* W23 = new float[trainingParams.W23_len];
	float* bias2 = new float[trainingParams.bias2_len];
	float* bias3 = new float[trainingParams.bias3_len];

	// Copy it back to neural network data structure
	err = hipMemcpy(W12, trainingParams.W12, trainingParams.W12_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(W23, trainingParams.W23, trainingParams.W23_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias2, trainingParams.bias2, trainingParams.bias2_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias3, trainingParams.bias3, trainingParams.bias3_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	// Free the cuda buffers
	hipFree (trainingParams.images);
	trainingParams.images = nullptr;
	hipFree (trainingParams.labels);
	trainingParams.labels = nullptr;
	hipFree (trainingParams.W12);
	trainingParams.W12 = nullptr;
	hipFree (trainingParams.W23);
	trainingParams.W23 = nullptr;
	hipFree (trainingParams.bias2);
	trainingParams.bias2 = nullptr;
	hipFree (trainingParams.bias3);
	trainingParams.bias3 = nullptr;
	hipFree (trainingParams.output2);
	trainingParams.output2 = nullptr;
	hipFree (trainingParams.output3);
	trainingParams.output3 = nullptr;

	//
	// Copy the weight data into the c++ data structure.
	//
	Layer* hidden = getLayer(HIDDEN);
	trainingParams.activationFunction2 = hidden->actFctType;
	for (size_t j = 0; j < hidden->nodes.size(); ++j) {
		Layer::Node* node = hidden->nodes[j];
		node->bias = bias2[j];
		for (size_t i = 0; i < node->weights.size(); ++i) {
			node->weights[i] = W12[j * hidden->nodes.size() + i];
		}
	}

	Layer* output = getLayer(OUTPUT);
	trainingParams.activationFunction3 = output->actFctType;
	for (size_t j = 0; j < output->nodes.size(); ++j) {
		Layer::Node* node = output->nodes[j];
		node->bias = bias3[j];
		for (size_t i = 0; i < node->weights.size(); ++i) {
			node->weights[i] = W23[j * output->nodes.size() + i];
		}
	}

	// Delete the host buffers
	delete[] W12;
	W12 = nullptr;
	delete[] W23;
	W23 = nullptr;
	delete[] bias2;
	bias2 = nullptr;
	delete[] bias3;
	bias3 = nullptr;
}

__device__ void d_print(GPUTrainingParameters const params) {
	printf("TrainingParams:\n"
			"  W12: %p\n"
		    "  W1_len: %lu\n"
			"  W2: %p\n"
			"  W2_len: %lu\n"
			"  errorThreshold: %f\n"
			"  width: %lu\n"
			"  height: %lu\n"
			"  numExamples: %lu\n"
			"  numHiddenNodes: %lu\n",
			params.W12,
			params.W12_len,
			params.W23,
			params.W23_len,
			params.errorThreshold,
			params.width,
			params.height,
			params.numExamples,
			params.numHiddenNodes);
}

__device__ void d_mul_shared(Matrix A, Matrix B, Matrix C);
__device__ void d_activate_layer(float* const, size_t const, NeuralNetwork::ActFctType);

__global__ void d_feed_forward(GPUTrainingParameters const params) {

	size_t const numImages = params.numHiddenNodes;

	Matrix W12;
	W12.rows = params.numHiddenNodes;
	W12.cols = params.width * params.height;
	W12.layout = Matrix::ROW_MAJOR;
	W12.data = params.W12; // Global data pointer
	if (W12.rows * W12.cols != params.W12_len) {
		printf("ERROR: W12 matrix has wrong dimensions: %lu x %lu != %lu\n", W12.rows, W12.cols, params.W12_len);
	}

	Matrix imgs;
	imgs.rows = params.width * params.height;
	imgs.cols = numImages;
	imgs.layout = Matrix::COLUMN_MAJOR;
	imgs.data = params.images; // Global data pointer, column major, yields one image in each column vector.

	Matrix hiddenOutput;
	hiddenOutput.rows = params.numHiddenNodes;
	hiddenOutput.cols = numImages;
	hiddenOutput.layout = Matrix::ROW_MAJOR;
	hiddenOutput.data = params.output2;
	if (hiddenOutput.rows * hiddenOutput.cols != params.output2_len) {
		printf("ERROR: HiddenOutput matrix has wrong dimensions: %lu x %lu != %lu\n", hiddenOutput.rows, hiddenOutput.cols, params.output2_len);
	}

	d_mul_shared(W12, imgs, hiddenOutput);
	d_activate_layer(params.output2, params.output2_len, params.activationFunction2);

	Matrix W23;
	W23.rows = NUM_DIGITS;
	W23.cols = params.numHiddenNodes;
	W23.layout = Matrix::ROW_MAJOR;
	W23.data = params.W23;
	if (W23.rows * W23.cols != params.W23_len) {
		printf("ERROR: W23 matrix has wrong dimensions: %lu x %lu != %lu\n", W23.rows, W23.cols, params.W23_len);
	}

	Matrix output;
	output.rows = W23.rows;
	output.cols = hiddenOutput.cols;
	output.layout = Matrix::ROW_MAJOR;
	output.data = params.output3;
	if (output.rows * output.cols != params.output3_len) {
		printf("ERROR: Output matrix has wrong dimensions: %lu x %lu != %lu\n", output.rows, output.cols, params.output3_len);
	}

	d_mul_shared(W23, hiddenOutput, output);
	d_activate_layer(params.output3, params.output3_len, params.activationFunction3);
}

__global__ void d_back_propagate(GPUTrainingParameters const params) {

}

__device__ void d_activate_layer(float* const data, size_t const len, NeuralNetwork::ActFctType functionType) {

	// Target index for this thread.
	size_t const idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y + blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	// If the target index would handle an element outside of the data buffer, terminate.
	if (idx >= len) {
		return;
	}

	switch (functionType) {
	case NeuralNetwork::SIGMOID:
		data[idx] = 1.0f / (1.0f + exp(-data[idx]));
		break;
	case NeuralNetwork::TANH:
		data[idx] = tanh(data[idx]);
		break;
	}
}

/**
 * Computes C = AB where the dimensions of A and be have to be a multiple of MATRIX_SIZE_DIVISOR.
 *
 * @param[in] A first factor of the matrix multiplication.
 * @param[in] B second factor of the multiplication.
 * @param[out] C Matrix holding the result. Must provide enough storage space.
 */
__device__ void d_mul_shared(Matrix A, Matrix B, Matrix C) {

	if (A.cols != B.rows) {

		printf("Incompatible matrices: (%lu, %lu) x (%lu, %lu)\n", A.rows, A.cols, B.rows, B.cols);
		return;
	}

	// Not needed anymore.
//	if (A.cols % MATRIX_SIZE_DIVISOR != 0 ||
//	    A.rows % MATRIX_SIZE_DIVISOR != 0 ||
//	    B.cols % MATRIX_SIZE_DIVISOR != 0 ||
//	    B.rows % MATRIX_SIZE_DIVISOR != 0) {
//
//		printf("Matrix dimensions not a multiple of %hu: (%lu, %lu) x (%lu, %lu)\n", MATRIX_SIZE_DIVISOR, A.rows, A.cols, B.rows, B.cols);
//		return;
//	}

	// The block caches are row major.
	__shared__ float blockCacheA[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];
	__shared__ float blockCacheB[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];

	// If this thread has nothing to do, because it would access invalid memory, exit
	if (blockIdx.x * MATRIX_SIZE_DIVISOR + threadIdx.x > C.cols ||
		blockIdx.y * MATRIX_SIZE_DIVISOR + threadIdx.y > C.rows) {
		return;
	}

	float threadValue = 0.0f;
	unsigned int const numSubBlocks = A.cols / MATRIX_SIZE_DIVISOR;
	for (int k = 0; k < numSubBlocks; ++k)
	{
		if (A.layout == Matrix::COLUMN_MAJOR) {
			blockCacheA[threadIdx.y][threadIdx.x] = A.data[(blockIdx.y + k * A.cols) * MATRIX_SIZE_DIVISOR + threadIdx.y + threadIdx.x * A.cols];
		} else if (A.layout == Matrix::ROW_MAJOR) {
			blockCacheA[threadIdx.y][threadIdx.x] = A.data[(blockIdx.y * A.cols + k) * MATRIX_SIZE_DIVISOR + threadIdx.y * A.cols + threadIdx.x];
		}

		if (B.layout == Matrix::COLUMN_MAJOR) {
			blockCacheB[threadIdx.y][threadIdx.x] = B.data[(blockIdx.x * B.cols + k) * MATRIX_SIZE_DIVISOR + threadIdx.y + threadIdx.x * B.cols];
		} else if (B.layout == Matrix::ROW_MAJOR) {
			blockCacheB[threadIdx.y][threadIdx.x] = B.data[(blockIdx.x + k * B.cols) * MATRIX_SIZE_DIVISOR + threadIdx.y * B.cols + threadIdx.x];
		}

		__syncthreads();

		#pragma unroll
		for (int i = 0; i < MATRIX_SIZE_DIVISOR; ++i)
		{
			threadValue += blockCacheA[threadIdx.y][i] * blockCacheB[i][threadIdx.x];
		}

		__syncthreads();
	}

	if (C.layout == Matrix::COLUMN_MAJOR) {
		C.data[(blockIdx.y + blockIdx.x * C.cols) * MATRIX_SIZE_DIVISOR + threadIdx.y + threadIdx.x * C.cols] = threadValue;
	} else if (C.layout == Matrix::ROW_MAJOR) {
		C.data[(blockIdx.y * C.cols + blockIdx.x) * MATRIX_SIZE_DIVISOR + threadIdx.y * C.cols + threadIdx.x] = threadValue;
	}
}
