#include "hip/hip_runtime.h"
#include "NeuralNetworkCUDA.h"

#include <iostream>
#include <cmath>

using namespace std;

__host__ NeuralNetworkCUDA::NeuralNetworkCUDA(const int inpCount,
		const int hidCount, const int outCount, const double learningRate) :
		NeuralNetwork(inpCount, hidCount, outCount, learningRate) {
}

__host__ NeuralNetworkCUDA::~NeuralNetworkCUDA() {
}

#define MATRIX_SIZE_DIVISOR 28
#define NUM_DIGITS 10

struct GPUTrainingParameters {
	/* Training data. */
	uint8_t* images;
	uint8_t* labels;

	/* Training data parameters. */
	size_t numExamples;
	size_t numHiddenNodes;
	size_t width;
	size_t height;

	/* Weight matrices. */
	float* W12;
	size_t W12_len;
	float* bias2;
	size_t bias2_len;
	float* W23;
	size_t W23_len;
	float* bias3;
	size_t bias3_len;

	/* Training parameters. */
	float errorThreshold;
	float maxDerivation;
};

struct GPUSharedMemoryLayout {
	size_t W1_pos = 0;
	size_t W1_size = 0;
	size_t W2_pos = 0;
	size_t W2_size = 0;
	size_t inputBias_pos = 0;
	size_t inputBias_size = 0;
	size_t hiddenOutput_pos = 0;
	size_t hiddenOutput_size = 0;
	size_t hiddenBias_pos = 0;
	size_t hiddenBias_size = 0;
	size_t outputOutput_pos = 0;
	size_t outputOutput_size = 0;
	size_t outputBias_pos = 0;
	size_t outputBias_size = 0;
	size_t image_pos = 0;
	size_t image_size = 0;
} gpuSharedMemoryLayout;

struct Matrix {
	size_t rows;
	size_t cols;
	float* data;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void trainCUDA(GPUTrainingParameters const);

__host__ void NeuralNetworkCUDA::train(MNISTImageDataset const& images,
		MNISTLableDataset const& labels, double const training_error_threshold,
		double const max_derivation) {

	if (images.size() <= 0)
		return;
	if (labels.size() <= 0)
		return;

	// Collect memory in RAM
	size_t const singleImgPixCount = images.front().total();
	size_t const allImgBufElements = singleImgPixCount * images.size();
	uint8_t* imgData = new uint8_t[allImgBufElements];
	uint8_t* it = imgData;
	for (cv::Mat const& img : images) {
		if (img.isContinuous()) {
			std::copy(img.datastart, img.dataend, it);
		} else {
			cerr << "cv::Mat is not continuous." << endl;
		}
		it += img.total() * img.elemSize();
	}

	hipError_t err;

	Layer* inputLayer  = getLayer(INPUT);
	Layer* hiddenLayer = getLayer(HIDDEN);
	Layer* outputLayer = getLayer(OUTPUT);

	GPUTrainingParameters trainingParams;
	trainingParams.numExamples = images.size();
	trainingParams.width = images.front().cols;
	trainingParams.height = images.front().rows;
	trainingParams.numHiddenNodes = hiddenLayer->nodes.size();
	trainingParams.errorThreshold = training_error_threshold;
	trainingParams.maxDerivation = max_derivation;

	//
	// Allocate cuda memory
	//

	// Images
	err = hipMalloc((void**) &trainingParams.images, allImgBufElements * sizeof(uint8_t));
	assert(err == hipSuccess);

	// Labels
	err = hipMalloc((void**) &trainingParams.labels, labels.size() * sizeof(uint8_t));
	assert(err == hipSuccess);

	// Storage for the first weight matrix
	trainingParams.W12_len = inputLayer->nodes.size() * hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W12, trainingParams.W12_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the hidden layer bias vector
	trainingParams.bias2_len = hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.bias2, trainingParams.bias2_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the second weight matrix
	trainingParams.W23_len = hiddenLayer->nodes.size() * outputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W23, trainingParams.W23_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer bias vector
	trainingParams.bias3_len = outputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.bias3, trainingParams.bias3_len * sizeof(float));
	assert(err == hipSuccess);

	//
	// Copy data to graphics card
	//
	err = hipMemcpy(trainingParams.images, imgData, allImgBufElements * sizeof(uint8_t), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.labels, labels.data(),labels.size() * sizeof(uint8_t), hipMemcpyHostToDevice);
	assert(err == hipSuccess);

	delete[] imgData;
	imgData = nullptr;

	// Configure Grid, i.e. setup Blocks and Threads
	dim3 numBlocks(1);
	dim3 threadsPerBlock(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
	cout << "Blocks:            (" << numBlocks.x << ", " << numBlocks.y << ")"
			<< endl;
	cout << "Threads per block: (" << threadsPerBlock.x << ", "
			<< threadsPerBlock.y << ")" << endl;

//	size_t sharedMemorySize = 0;

	// Size of the first weight matrix
//	gpuSharedMemoryLayout.W1_pos = 0;
//	gpuSharedMemoryLayout.W1_size = trainingParams.W1_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W1_size;
//
//	// Size of the second weight matrix
//	gpuSharedMemoryLayout.W2_pos = gpuSharedMemoryLayout.W1_pos + gpuSharedMemoryLayout.W1_size;
//	gpuSharedMemoryLayout.W2_size = trainingParams.W2_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W2_size;
//
//	// Size of the hidden layer output nodes
//	gpuSharedMemoryLayout.hiddenOutput_pos = gpuSharedMemoryLayout.W2_pos + gpuSharedMemoryLayout.W2_size;
//	gpuSharedMemoryLayout.hiddenOutput_size = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenOutput_size;
//
//	// Size of the output layer output values
//	gpuSharedMemoryLayout.outputOutput_pos = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.outputOutput_size = outputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.outputOutput_size;
//
//	// Size of the hidden bias vector
//	gpuSharedMemoryLayout.hiddenBias_pos   = gpuSharedMemoryLayout.outputOutput_pos + gpuSharedMemoryLayout.outputOutput_size;
//	gpuSharedMemoryLayout.hiddenBias_size  = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenBias_size;
//
//	// Size of the input bias vector
//	gpuSharedMemoryLayout.inputBias_pos    = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.inputBias_size   = inputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.inputBias_size;
//
//	// Size of the input vector
//	gpuSharedMemoryLayout.image_pos        = gpuSharedMemoryLayout.inputBias_pos + gpuSharedMemoryLayout.inputBias_size;
//	gpuSharedMemoryLayout.image_size       = inputLayer->nodes.size() * sizeof(uint8_t);
//	sharedMemorySize += gpuSharedMemoryLayout.image_size;

	hipMemset(trainingParams.W12, 0.0, trainingParams.W12_len * sizeof(float));
	hipMemset(trainingParams.W23, 0.0, trainingParams.W23_len * sizeof(float));
	hipMemset(trainingParams.bias2, 0.0, trainingParams.bias2_len * sizeof(float));
	hipMemset(trainingParams.bias3, 0.0, trainingParams.bias3_len * sizeof(float));

	// Call graphics card functions
	trainCUDA<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	//
	// Retreive the data
	//
	float* W12 = new float[trainingParams.W12_len];
	float* W23 = new float[trainingParams.W23_len];
	float* bias2 = new float[trainingParams.bias2_len];
	float* bias3 = new float[trainingParams.bias3_len];

	// Copy it back to neural network data structure
	err = hipMemcpy(W12, trainingParams.W12, trainingParams.W12_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(W23, trainingParams.W23, trainingParams.W23_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias2, trainingParams.bias2, trainingParams.bias2_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias3, trainingParams.bias3, trainingParams.bias3_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	// Free the cuda buffers
	hipFree (trainingParams.images);
	trainingParams.images = nullptr;
	hipFree (trainingParams.labels);
	trainingParams.labels = nullptr;
	hipFree (trainingParams.W12);
	trainingParams.W12 = nullptr;
	hipFree (trainingParams.W23);
	trainingParams.W23 = nullptr;
	hipFree (trainingParams.bias2);
	trainingParams.bias2 = nullptr;
	hipFree (trainingParams.bias3);
	trainingParams.bias3 = nullptr;

	//
	// Copy the weight data into the c++ data structure.
	//
	Layer* hidden = getLayer(HIDDEN);
	for (size_t j = 0; j < hidden->nodes.size(); ++j) {
		Layer::Node* node = hidden->nodes[j];
		node->bias = bias2[j];
		for (size_t i = 0; i < node->weights.size(); ++i) {
			node->weights[i] = W12[j * hidden->nodes.size() + i];
		}
	}

	Layer* output = getLayer(OUTPUT);
	for (size_t j = 0; j < output->nodes.size(); ++j) {
		Layer::Node* node = output->nodes[j];
		node->bias = bias3[j];
		for (size_t i = 0; i < node->weights.size(); ++i) {
			node->weights[i] = W23[j * output->nodes.size() + i];
		}
	}

	// Delete the host buffers
	delete[] W12;
	W12 = nullptr;
	delete[] W23;
	W23 = nullptr;
	delete[] bias2;
	bias2 = nullptr;
	delete[] bias3;
	bias3 = nullptr;
}

__device__ void printCuda(GPUTrainingParameters const params) {
	printf("TrainingParams:\n"
			"  W12: %p\n"
		    "  W1_len: %lu\n"
			"  W2: %p\n"
			"  W2_len: %lu\n"
			"  errorThreshold: %f\n"
			"  width: %lu\n"
			"  height: %lu\n"
			"  numExamples: %lu\n"
			"  numHiddenNodes: %lu\n",
			params.W12,
			params.W12_len,
			params.W23,
			params.W23_len,
			params.errorThreshold,
			params.width,
			params.height,
			params.numExamples,
			params.numHiddenNodes);
}

__device__ void feedForward(GPUTrainingParameters const);
__device__ void backPropagate(float sharedMem[]);

__global__ void trainCUDA(GPUTrainingParameters const params) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printCuda(params);
	}
	//
	// Initialize the internal weight matrices for each network.
	//

	// Weight matrices will be column major for better bank access.
	//extern __shared__ float sharedMem[];


	// The layer outputs will be stored in shared vectors.

	//
	// Start training
	//
	bool needsFurtherTraining = true;
	float error = 100000000.0f;
	while (needsFurtherTraining) {

		float newError = 0;

		//NeuralNetworkParallel nnp_merge(*this);

//		{
//			//NeuralNetworkParallel nnp_local(*this);
//			size_t localErrCount = 0;
//
//			for (size_t imgCount = 0; imgCount < images.size(); imgCount++) {
//				// Convert the MNIST image to a standardized vector format and feed into the network
//				nnp_local.feedInput(images[imgCount]);
//
//				// Feed forward all layers (from input to hidden to output) calculating all nodes' output
				//feedForward(params);

				// Back propagate the error and adjust weights in all layers accordingly
				//backPropagate(nullptr);
//
//				// Classify image by choosing output cell with highest output
//				int classification = nnp_local.getNetworkClassification();
//				if (classification != labels[imgCount])
//					localErrCount++;
//
//				// Display progress during training
//				if ((imgCount % every_ten_percent) == 0) {
//					cout << "x";
//					cout.flush();
//				}
//			}
//
//			newError += static_cast<double>(localErrCount) / static_cast<double>(images.size());
//
//			// merge network weights together
//			mergeNeuralNetworks(nnp_local, nnp_merge, this);
//
//			//cout << "Thread-" << omp_get_thread_num() << ": Error=" << localErrCount << ", Images=" << localImageProcessed << endl;
//		}
//
//		mergeNeuralNetworks(nnp_merge, *this, this);

		if (newError < error) {
			error = newError;
		}

		needsFurtherTraining = !(error < params.errorThreshold || newError > error + params.maxDerivation);

//		cout << " Error: " << error * 100.0 << "%, NewError: " << newError * 100.0 << "%" << endl;
	}
}

__device__ void d_mul_shared(Matrix A, Matrix B, Matrix C);

__device__ void feedForward(GPUTrainingParameters const params) {

	__shared__ float* hiddenOutputs[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];
	__shared__ float* outputs[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];
	__shared__ float* imageData[MATRIX_SIZE_DIVISOR * MATRIX_SIZE_DIVISOR];
	__shared__ float* alignedW2[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];

	size_t const numImages = params.numHiddenNodes;

	Matrix W1;
	W1.rows = params.numHiddenNodes;
	W1.cols = params.width * params.height;
	W1.data = params.W12;

	Matrix imgs;
	imgs.rows = params.width * params.height;
	imgs.cols = numImages;
	imgs.data = (float*)imageData;
	imgs.data[threadIdx.x * threadIdx.y] = params.images[threadIdx.x * threadIdx.y];

	Matrix foobar;
	foobar.rows = params.numHiddenNodes;
	foobar.cols = numImages;
	foobar.data = (float*)hiddenOutputs;

	d_mul_shared(W1, imgs, foobar);

	Matrix W2;
	W2.rows = foobar.rows;
	W2.cols = params.numHiddenNodes;
	W2.data = (float*)alignedW2;
	//memcpy(W2.data, params.W2, params.W2_len * sizeof(float));
	//W23.data = params.W23;

	Matrix O;
	O.rows = W2.rows;
	O.cols = numImages;
	O.data = (float*)outputs;

	//d_mul_shared(W2, foobar, O);

	//delete[] imgs.data;
	//delete[] W23.data;
}

__device__ void backPropagate(float sharedMem[]) {

}

/**
 * Computes C = AB where the dimensions of A and be have to be a multiple of MATRIX_SIZE_DIVISOR.
 *
 * Matrices are expected to be row-major.
 *
 * @param[in] A first factor of the matrix multiplication.
 * @param[in] B second factor of the multiplication.
 * @param[out] C Matrix holding the result. Must provide enough storage space.
 */
__device__ void d_mul_shared(Matrix A, Matrix B, Matrix C) {

	if (A.cols != B.rows) {

		printf("Invalid matrix sizes: (%lu, %lu)x(%lu, %lu)\n", A.rows, A.cols, B.rows, B.cols);
		return;
	}

	if (A.cols % MATRIX_SIZE_DIVISOR != 0 ||
	    A.rows % MATRIX_SIZE_DIVISOR != 0 ||
	    B.cols % MATRIX_SIZE_DIVISOR != 0 ||
	    B.rows % MATRIX_SIZE_DIVISOR != 0) {

		printf("Matrix dimensions not a multiple of %hu: (%lu, %lu)x(%lu, %lu)\n", MATRIX_SIZE_DIVISOR, A.rows, A.cols, B.rows, B.cols);
		return;
	}

	__shared__ float blockCacheA[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];
	__shared__ float blockCacheB[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];

	// Column major!
	float threadValue = 0.0f;
	unsigned int const numSubBlocks = A.cols / MATRIX_SIZE_DIVISOR;
	for (int k = 0; k < numSubBlocks; ++k)
	{
		blockCacheA[threadIdx.x][threadIdx.y] = A.data[k * MATRIX_SIZE_DIVISOR + threadIdx.y * A.cols + threadIdx.x];
		printf("idx: %lu\n", k * B.cols * MATRIX_SIZE_DIVISOR + threadIdx.y * B.cols + threadIdx.x);
		blockCacheB[threadIdx.y][threadIdx.x] = B.data[k * B.cols * MATRIX_SIZE_DIVISOR + threadIdx.y * B.cols + threadIdx.x];

		__syncthreads();

	#pragma unroll
		for (int i = 0; i < MATRIX_SIZE_DIVISOR; ++i)
		{
			threadValue += blockCacheA[i][threadIdx.y] * blockCacheB[i][threadIdx.x];
		}

		__syncthreads();
	}

	C.data[threadIdx.y * C.cols + threadIdx.x] = threadValue;
}
