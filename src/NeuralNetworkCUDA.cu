#include "hip/hip_runtime.h"
#include "NeuralNetworkCUDA.h"

#include <iostream>

using namespace std;

__host__ NeuralNetworkCUDA::NeuralNetworkCUDA(const int inpCount,
		const int hidCount, const int outCount, const double learningRate) :
		NeuralNetwork(inpCount, hidCount, outCount, learningRate) {
}

__host__ NeuralNetworkCUDA::~NeuralNetworkCUDA() {
}

struct GPUTrainingParameters {
	/* Training data. */
	uint8_t* images;
	uint8_t* labels;

	/* Training data parameters. */
	size_t numExamples;
	size_t numHiddenNodes;
	size_t width;
	size_t height;

	/* Weight matrices. */
	float* W1;
	size_t W1_len;
	float* W2;
	size_t W2_len;

	/* Training parameters. */
	float errorThreshold;
};

__global__ void trainCUDA(GPUTrainingParameters const);

__host__ void NeuralNetworkCUDA::train(MNISTImageDataset const& images,
		MNISTLableDataset const& labels, double const training_error_threshold,
		double const max_derivation) {

	if (images.size() <= 0)
		return;
	if (labels.size() <= 0)
		return;

	// Collect memory in RAM
	size_t const singleImgPixCount = images.front().total();
	size_t const allImgBufElements = singleImgPixCount * images.size();
	uint8_t* imgData = new uint8_t[allImgBufElements];
	uint8_t* it = imgData;
	for (cv::Mat const& img : images) {
		if (img.isContinuous()) {
			std::copy(img.datastart, img.dataend, it);
		} else {
			cerr << "cv::Mat is not continuous." << endl;
		}
		it += img.total() * img.elemSize();
	}

	hipError_t err;

	Layer* inputLayer  = getLayer(INPUT);
	Layer* hiddenLayer = getLayer(HIDDEN);
	Layer* outputLayer = getLayer(OUTPUT);

	GPUTrainingParameters trainingParams;
	trainingParams.numExamples = images.size();
	trainingParams.width = images.front().cols;
	trainingParams.height = images.front().rows;
	trainingParams.numHiddenNodes = hiddenLayer->nodes.size();
	trainingParams.errorThreshold = training_error_threshold;

	//
	// Allocate cuda memory
	//

	// Images
	err = hipMalloc((void**) &trainingParams.images,
			allImgBufElements * sizeof(uint8_t));
	assert(err == hipSuccess);

	// Labels
	err = hipMalloc((void**) &trainingParams.labels,
			labels.size() * sizeof(uint8_t));
	assert(err == hipSuccess);

	// Storage for the first weight matrix
	trainingParams.W1_len = inputLayer->nodes.size() * hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W1, trainingParams.W1_len * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the first weight matrix
	trainingParams.W2_len = hiddenLayer->nodes.size() * outputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W2, trainingParams.W2_len * sizeof(float));
	assert(err == hipSuccess);

	//
	// Copy data to graphics card
	//
	err = hipMemcpy(trainingParams.images, imgData,
			allImgBufElements * sizeof(uint8_t), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.labels, labels.data(),
			labels.size() * sizeof(uint8_t), hipMemcpyHostToDevice);

	delete[] imgData;
	imgData = nullptr;

	// Configure Grid, i.e. setup Blocks and Threads
	dim3 numBlocks(32, 32);
	dim3 threadsPerBlock(16, 16);
	cout << "Blocks:            (" << numBlocks.x << ", " << numBlocks.y << ")"
			<< endl;
	cout << "Threads per block: (" << threadsPerBlock.x << ", "
			<< threadsPerBlock.y << ")" << endl;

	// Call graphics card functions
	trainCUDA<<<numBlocks, threadsPerBlock>>>(trainingParams);

	//
	// Retreive the data
	//

	float* W1 = new float[trainingParams.W1_len];
	float* W2 = new float[trainingParams.W2_len];

	// Copy it back to neural network data structure
	err = hipMemcpy(W1, trainingParams.W1, trainingParams.W1_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(W2, trainingParams.W2, trainingParams.W2_len * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	// Free the cuda buffers
	hipFree (trainingParams.images);
	trainingParams.images = nullptr;
	hipFree (trainingParams.labels);
	trainingParams.labels = nullptr;
	hipFree (trainingParams.W1);
	trainingParams.W1 = nullptr;
	hipFree (trainingParams.W2);
	trainingParams.W2 = nullptr;

	//
	// Copy the weight data into the c++ data structure.
	//

	// Delete the host buffers
	delete[] W1;
	W1 = nullptr;
	delete[] W2;
	W2 = nullptr;
}

__global__ void trainCUDA(GPUTrainingParameters const params) {

}
