#include "hip/hip_runtime.h"
#include "NeuralNetworkCUDA.h"

#include <iostream>
#include <cmath>

using namespace std;

__host__ NeuralNetworkCUDA::NeuralNetworkCUDA(const int inpCount,
		const int hidCount, const int outCount, const double learningRate) :
		NeuralNetwork(inpCount, hidCount, outCount, learningRate) {
}

__host__ NeuralNetworkCUDA::~NeuralNetworkCUDA() {
}

#define MATRIX_SIZE_DIVISOR 28
#define NUM_DIGITS 10

struct Matrix {
	enum Layout {
		ROW_MAJOR,
		COLUMN_MAJOR
	};

	size_t rows;
	size_t cols;
	Layout layout = ROW_MAJOR;
	float* data;
};

struct GPUTrainingParameters {
	/* Training data. */
	float* images;
	float* labels;

	/* Training data parameters. */
	size_t numExamples;
	size_t numHiddenNodes;
	size_t width;
	size_t height;
	// Number of images per training
	size_t batchSize;

	/* Weight matrices. */
	Matrix W12;
	Matrix W23;

	/* Biases */
	Matrix bias2;
	Matrix bias3;

	/* Layer data */
	Matrix output2;
	Matrix output3;

	NeuralNetwork::ActFctType activationFunction2;
	NeuralNetwork::ActFctType activationFunction3;

	/* Training parameters. */
	float errorThreshold;
	float maxDerivation;

	/* Temporary buffers, e.g. for back propagation. */
	Matrix tmp1;
	Matrix tmp2;
};

struct GPUSharedMemoryLayout {
	size_t W1_pos = 0;
	size_t W1_size = 0;
	size_t W2_pos = 0;
	size_t W2_size = 0;
	size_t inputBias_pos = 0;
	size_t inputBias_size = 0;
	size_t hiddenOutput_pos = 0;
	size_t hiddenOutput_size = 0;
	size_t hiddenBias_pos = 0;
	size_t hiddenBias_size = 0;
	size_t outputOutput_pos = 0;
	size_t outputOutput_size = 0;
	size_t outputBias_pos = 0;
	size_t outputBias_size = 0;
	size_t image_pos = 0;
	size_t image_size = 0;
} gpuSharedMemoryLayout;

__device__ float* d_matrix_pget(Matrix const M, size_t const y, size_t const x) {
	if (M.layout == Matrix::ROW_MAJOR) {
		return M.data + (x + y * M.cols);
	} else {
		return M.data + (x * M.rows + y);
	}
}

__device__ float d_matrix_get(Matrix const M, size_t const y, size_t const x) {
	return *d_matrix_pget(M, y, x);
}

__device__ void d_matrix_set(Matrix const M, size_t const y, size_t const x, float const value) {
	if (M.layout == Matrix::ROW_MAJOR) {
		M.data[x + y * M.cols] = value;
	} else {
		M.data[x * M.rows + y] = value;
	}
}

__device__ size_t d_matrix_size(Matrix const A) {
	return A.rows * A.cols;
}

size_t matrix_size(Matrix const A) {
	return A.rows * A.cols;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void d_feed_forward(GPUTrainingParameters const);
__global__ void d_back_propagate(GPUTrainingParameters const);

__host__ void NeuralNetworkCUDA::train(MNISTImageDataset const& images,
		MNISTLableDataset const& labels, double const training_error_threshold,
		double const max_derivation) {

	if (images.size() <= 0)
		return;
	if (labels.size() <= 0)
		return;

	Layer* const inputLayer  = getLayer(INPUT);
	Layer* const hiddenLayer = getLayer(HIDDEN);
	Layer* const outputLayer = getLayer(OUTPUT);

	GPUTrainingParameters trainingParams;
	trainingParams.numExamples = images.size();
	trainingParams.width = images.front().cols;
	trainingParams.height = images.front().rows;
	trainingParams.numHiddenNodes = hiddenLayer->nodes.size();
	trainingParams.errorThreshold = training_error_threshold;
	trainingParams.maxDerivation = max_derivation;
	trainingParams.batchSize = MATRIX_SIZE_DIVISOR;

	hipError_t err;

	//
	// Allocate cuda memory
	//
	size_t const singleImgPixCount = images.front().total();
	size_t const allImgBufElements = singleImgPixCount * images.size();

	// Images
	err = hipMalloc((void**) &trainingParams.images, allImgBufElements * sizeof(float));
	assert(err == hipSuccess);

	// Labels
	err = hipMalloc((void**) &trainingParams.labels, labels.size() * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the first weight matrix
	trainingParams.W12.rows = hiddenLayer->nodes.size();
	trainingParams.W12.cols = inputLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W12.data, matrix_size(trainingParams.W12) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the hidden layer bias vector
	trainingParams.bias2.rows = hiddenLayer->nodes.size();
	trainingParams.bias2.cols = 1;
	err = hipMalloc((void**) &trainingParams.bias2.data, matrix_size(trainingParams.bias2) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the second weight matrix
	trainingParams.W23.rows = outputLayer->nodes.size();
	trainingParams.W23.cols = hiddenLayer->nodes.size();
	err = hipMalloc((void**) &trainingParams.W23.data, matrix_size(trainingParams.W23) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer bias vector
	trainingParams.bias3.rows = outputLayer->nodes.size();
	trainingParams.bias3.cols = 1;
	err = hipMalloc((void**) &trainingParams.bias3.data, matrix_size(trainingParams.bias3) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer output vectors
	trainingParams.output2.rows = trainingParams.numHiddenNodes;
	trainingParams.output2.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.output2.data, matrix_size(trainingParams.output2) * sizeof(float));
	assert(err == hipSuccess);

	// Storage for the output layer output vectors
	trainingParams.output3.rows = outputLayer->nodes.size();
	trainingParams.output3.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.output3.data, matrix_size(trainingParams.output3) * sizeof(float));
	assert(err == hipSuccess);

	// Temporary storage of the size of the output vectors
	trainingParams.tmp1.rows = outputLayer->nodes.size();
	trainingParams.tmp1.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.tmp1.data, matrix_size(trainingParams.tmp1) * sizeof(float));
	assert(err == hipSuccess);

	// Temporary storage of the size of the output vectors
	trainingParams.tmp2.rows = outputLayer->nodes.size();
	trainingParams.tmp2.cols = trainingParams.batchSize;
	err = hipMalloc((void**) &trainingParams.tmp2.data, matrix_size(trainingParams.tmp2) * sizeof(float));
	assert(err == hipSuccess);

	//
	// Collect memory in RAM
	//
	float* imgData = new float[allImgBufElements];
	float* dst = imgData;
	for (cv::Mat const& img : images) {
		for (uint8_t* src = img.datastart; src != img.dataend;) {
			*(dst++) = static_cast<float>(*(src++));
		}
	}

	float* flabels = new float[labels.size()];
	dst = flabels;
	for (uint8_t const& l : labels) {
		*(dst++) = static_cast<float>(l);
	}

	float* W12 = new float[matrix_size(trainingParams.W12)];
	float* W23 = new float[matrix_size(trainingParams.W23)];
	float* bias2 = new float[matrix_size(trainingParams.bias2)];
	float* bias3 = new float[matrix_size(trainingParams.bias3)];

	//
	// Collect the initial weights and biases in buffers for submission to the GPU.
	//
	trainingParams.activationFunction2 = hiddenLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < hiddenLayer->nodes.size(); ++j) {
			Layer::Node* node = hiddenLayer->nodes[j];
			bias2[j] = node->bias;
			for (size_t i = 0; i < node->weights.size(); ++i) {
				W12[k] = node->weights[i];
				++k;
			}
		}
	}

	trainingParams.activationFunction3 = outputLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < outputLayer->nodes.size(); ++j) {
			Layer::Node* node = outputLayer->nodes[j];
			bias3[j] = node->bias;
			for (size_t i = 0; i < node->weights.size(); ++i) {
				W23[k] = node->weights[i];
				++k;
			}
		}
	}

	//
	// Copy data to graphics card
	//
	err = hipMemcpy(trainingParams.images, imgData, allImgBufElements * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.labels, flabels, labels.size() * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.W12.data, W12, matrix_size(trainingParams.W12) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.bias2.data, bias2, matrix_size(trainingParams.bias2) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.W23.data, W23, matrix_size(trainingParams.W23) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);
	err = hipMemcpy(trainingParams.bias3.data, bias3, matrix_size(trainingParams.bias3) * sizeof(float), hipMemcpyHostToDevice);
	assert(err == hipSuccess);

	delete[] imgData;
	imgData = nullptr;
	delete[] flabels;
	flabels = nullptr;

//	size_t sharedMemorySize = 0;

	// Size of the first weight matrix
//	gpuSharedMemoryLayout.W1_pos = 0;
//	gpuSharedMemoryLayout.W1_size = trainingParams.W1_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W1_size;
//
//	// Size of the second weight matrix
//	gpuSharedMemoryLayout.W2_pos = gpuSharedMemoryLayout.W1_pos + gpuSharedMemoryLayout.W1_size;
//	gpuSharedMemoryLayout.W2_size = trainingParams.W2_len * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.W2_size;
//
//	// Size of the hidden layer output nodes
//	gpuSharedMemoryLayout.hiddenOutput_pos = gpuSharedMemoryLayout.W2_pos + gpuSharedMemoryLayout.W2_size;
//	gpuSharedMemoryLayout.hiddenOutput_size = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenOutput_size;
//
//	// Size of the output layer output values
//	gpuSharedMemoryLayout.outputOutput_pos = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.outputOutput_size = outputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.outputOutput_size;
//
//	// Size of the hidden bias vector
//	gpuSharedMemoryLayout.hiddenBias_pos   = gpuSharedMemoryLayout.outputOutput_pos + gpuSharedMemoryLayout.outputOutput_size;
//	gpuSharedMemoryLayout.hiddenBias_size  = hiddenLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.hiddenBias_size;
//
//	// Size of the input bias vector
//	gpuSharedMemoryLayout.inputBias_pos    = gpuSharedMemoryLayout.hiddenOutput_pos + gpuSharedMemoryLayout.hiddenOutput_size;
//	gpuSharedMemoryLayout.inputBias_size   = inputLayer->nodes.size() * sizeof(float);
//	sharedMemorySize += gpuSharedMemoryLayout.inputBias_size;
//
//	// Size of the input vector
//	gpuSharedMemoryLayout.image_pos        = gpuSharedMemoryLayout.inputBias_pos + gpuSharedMemoryLayout.inputBias_size;
//	gpuSharedMemoryLayout.image_size       = inputLayer->nodes.size() * sizeof(uint8_t);
//	sharedMemorySize += gpuSharedMemoryLayout.image_size;

	// Configure Grid, i.e. setup Blocks and Threads
	dim3 numBlocks(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
	dim3 threadsPerBlock(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);
	cout << "Blocks:            (" << numBlocks.x << ", " << numBlocks.y << ")"
			<< endl;
	cout << "Threads per block: (" << threadsPerBlock.x << ", "
			<< threadsPerBlock.y << ")" << endl;

	// Call graphics card functions
	d_feed_forward<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	d_back_propagate<<<numBlocks, threadsPerBlock>>>(trainingParams);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	//
	// Retreive the data
	//

	// Copy it back to neural network data structure
	err = hipMemcpy(W12, trainingParams.W12.data, matrix_size(trainingParams.W12) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(W23, trainingParams.W23.data, matrix_size(trainingParams.W23) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias2, trainingParams.bias2.data, matrix_size(trainingParams.bias2) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(bias3, trainingParams.bias3.data, matrix_size(trainingParams.bias3) * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	// Free the cuda buffers
	hipFree (trainingParams.images);
	trainingParams.images = nullptr;
	hipFree (trainingParams.labels);
	trainingParams.labels = nullptr;
	hipFree (trainingParams.W12.data);
	trainingParams.W12.data = nullptr;
	hipFree (trainingParams.W23.data);
	trainingParams.W23.data = nullptr;
	hipFree (trainingParams.bias2.data);
	trainingParams.bias2.data = nullptr;
	hipFree (trainingParams.bias3.data);
	trainingParams.bias3.data = nullptr;
	hipFree (trainingParams.output2.data);
	trainingParams.output2.data = nullptr;
	hipFree (trainingParams.output3.data);
	trainingParams.output3.data = nullptr;
	hipFree (trainingParams.tmp1.data);
	trainingParams.tmp1.data = nullptr;
	hipFree (trainingParams.tmp2.data);
	trainingParams.tmp2.data = nullptr;

	//
	// Copy the weight data into the c++ data structure.
	//
	trainingParams.activationFunction2 = hiddenLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < hiddenLayer->nodes.size(); ++j) {
			Layer::Node* node = hiddenLayer->nodes[j];
			node->bias = bias2[j];
			for (size_t i = 0; i < node->weights.size(); ++i) {
				node->weights[i] = W12[k];
				++k;
			}
		}
	}

	trainingParams.activationFunction3 = outputLayer->actFctType;
	{
		size_t k = 0;
		for (size_t j = 0; j < outputLayer->nodes.size(); ++j) {
			Layer::Node* node = outputLayer->nodes[j];
			node->bias = bias3[j];
			for (size_t i = 0; i < node->weights.size(); ++i) {
				node->weights[i] = W23[k];
				++k;
			}
		}
	}

	// Delete the host buffers
	delete[] W12;
	W12 = nullptr;
	delete[] W23;
	W23 = nullptr;
	delete[] bias2;
	bias2 = nullptr;
	delete[] bias3;
	bias3 = nullptr;
}

__device__ void d_print(GPUTrainingParameters const params) {
	printf("TrainingParams:\n"
			"  W12: %p\n"
		    "  W1_len: %lu\n"
			"  W2: %p\n"
			"  W2_len: %lu\n"
			"  errorThreshold: %f\n"
			"  width: %lu\n"
			"  height: %lu\n"
			"  numExamples: %lu\n"
			"  numHiddenNodes: %lu\n",
			params.W12.data,
			d_matrix_size(params.W12),
			params.W23.data,
			d_matrix_size(params.W23),
			params.errorThreshold,
			params.width,
			params.height,
			params.numExamples,
			params.numHiddenNodes);
}

/* Matrix manipulation operations. */
__device__ void d_mul_base(Matrix C, Matrix const A, Matrix const B, void(*op)(float*, float const, float const));
__device__ void d_mul(Matrix C, Matrix const A, Matrix const B);
__device__ void d_mul_add(Matrix C, Matrix const A, Matrix const B);
__device__ void d_cwise_op(Matrix C, Matrix const A, Matrix const B, void(*op)(float*, float const, float const));
__device__ void d_cwise_mul(Matrix C, Matrix const A, Matrix const B);
__device__ void d_cwise_sub(Matrix C, Matrix const A, Matrix const B);

/* Neural network operations. */
__device__ void d_apply_activation(Matrix, NeuralNetwork::ActFctType);
__device__ void d_apply_activation_derivative(Matrix, NeuralNetwork::ActFctType);
__device__ void d_back_propagate_output(GPUTrainingParameters const);
__device__ void d_back_propagate_hidden(GPUTrainingParameters const);
__device__ void d_fill_target_output(GPUTrainingParameters const, Matrix);
__device__ void d_set_bias(Matrix output, Matrix const bias);
__device__ void d_fill_random(Matrix);

__global__ void d_feed_forward(GPUTrainingParameters const params) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_feed_forward\n");
	}

//	Matrix W12;
//	W12.rows = params.numHiddenNodes;
//	W12.cols = params.width * params.height;
//	W12.layout = Matrix::ROW_MAJOR;
//	W12.data = params.W12; // Global data pointer
//	if (W12.rows * W12.cols != params.W12_len) {
//		printf("ERROR: W12 matrix has wrong dimensions: %lu x %lu != %lu\n", W12.rows, W12.cols, params.W12_len);
//	}

	Matrix imgs;
	imgs.rows = params.width * params.height;
	imgs.cols = params.batchSize;
	imgs.layout = Matrix::COLUMN_MAJOR;
	imgs.data = params.images; // Global data pointer, column major, yields one image in each column vector.

//	Matrix hiddenOutput;
//	hiddenOutput.rows = params.numHiddenNodes;
//	hiddenOutput.cols = params.batchSize;
//	hiddenOutput.layout = Matrix::ROW_MAJOR;
//	hiddenOutput.data = params.output2;
//	if (hiddenOutput.rows * hiddenOutput.cols != params.output2_len) {
//		printf("ERROR: HiddenOutput matrix has wrong dimensions: %lu x %lu != %lu\n", hiddenOutput.rows, hiddenOutput.cols, params.output2_len);
//	}

//	Matrix bias2;
//	bias2.rows = params.numHiddenNodes;
//	bias2.cols = 1;
//	bias2.data = params.bias2;
//	if (bias2.rows * bias2.cols != params.bias2_len) {
//		printf("ERROR: Bias2 has wrong dimensions: %lu x %lu != %lu\n", bias2.rows, bias2.cols, params.bias2_len);
//	}

	d_set_bias(params.output2, params.bias2);
	d_mul_add(params.output2, params.W12, imgs);
	d_apply_activation(params.output2, params.activationFunction2);

//	Matrix W23;
//	W23.rows = NUM_DIGITS;
//	W23.cols = params.numHiddenNodes;
//	W23.layout = Matrix::ROW_MAJOR;
//	W23.data = params.W23;
//	if (W23.rows * W23.cols != params.W23_len) {
//		printf("ERROR: W23 matrix has wrong dimensions: %lu x %lu != %lu\n", W23.rows, W23.cols, params.W23_len);
//	}

//	Matrix output;
//	output.rows = NUM_DIGITS;
//	output.cols = params.batchSize;
//	output.layout = Matrix::ROW_MAJOR;
//	output.data = params.output3;
//	if (output.rows * output.cols != params.output3_len) {
//		printf("ERROR: Output matrix has wrong dimensions: %lu x %lu != %lu\n", output.rows, output.cols, params.output3_len);
//	}

//	Matrix bias3;
//	bias3.rows = NUM_DIGITS;
//	bias3.cols = 1;
//	bias3.data = params.bias3;
//	if (bias3.rows * bias3.cols != params.bias3_len) {
//		printf("ERROR: Bias3 has wrong dimensions: %lu x %lu != %lu\n", bias3.rows, bias3.cols, params.bias3_len);
//	}

	d_set_bias(params.output3, params.bias3);
	d_mul_add(params.output3, params.W23, params.output2);
	d_apply_activation(params.output3, params.activationFunction3);

//	d_fill_random(W12);
//	d_fill_random(W23);
//	d_fill_random(bias2);
//	d_fill_random(bias3);
}

__global__ void d_back_propagate(GPUTrainingParameters const params) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_back_propagate\n");
	}

	d_back_propagate_output(params);
	d_back_propagate_hidden(params);
}

__device__ void d_back_propagate_output(GPUTrainingParameters const params) {

//	Matrix targetOutput;
//	targetOutput.rows = NUM_DIGITS;
//	targetOutput.cols = params.batchSize;
//	targetOutput.data = params.tmp1;
//	if (targetOutput.rows * targetOutput.cols != params.tmp1_len) {
//		printf("d_back_propagate_output: targetOutput matrix has wrong dimensions: %lu x %lu != %lu\n", targetOutput.rows, targetOutput.cols, params.tmp1_len);
//	}
	Matrix targetOutput = params.tmp1;

	// Compute the target output based on the labels
	d_fill_target_output(params, targetOutput);

//	Matrix output;
//	output.rows = NUM_DIGITS;
//	output.cols = params.batchSize;
//	output.data = params.output3;
//	if (output.rows * output.cols != params.output3_len) {
//		printf("d_back_propagate_output: Output matrix has wrong dimensions: %lu x %lu != %lu\n", output.rows, output.cols, params.output3_len);
//	}

	// Save the difference into the target output buffer
	Matrix difference = targetOutput;
	// Reuse the output buffer for saving the error, for now. Perhaps this is a problem later on.
	Matrix error = params.output3;

	d_cwise_sub(difference, targetOutput, params.output3);
	d_apply_activation_derivative(params.output3, params.activationFunction3);
	d_cwise_mul(error, params.output3, difference);

//	Matrix hiddenOutput;
//	hiddenOutput.rows = params.batchSize;
//	hiddenOutput.cols = params.numHiddenNodes;
//	hiddenOutput.layout = Matrix::ROW_MAJOR;
//	hiddenOutput.data = params.output2;
//	if (hiddenOutput.rows * hiddenOutput.cols != params.output2_len) {
//		printf("d_back_propagate_output: hidden output matrix has wrong dimensions: %lu x %lu != %lu\n", hiddenOutput.rows, hiddenOutput.cols, params.output2_len);
//	}

//	Matrix W23;
//	W23.rows = NUM_DIGITS;
//	W23.cols = params.numHiddenNodes;
//	W23.data = params.W23;
//	if (W23.rows * W23.cols != params.W23_len) {
//		printf("d_back_propagate_output: W23 matrix has wrong dimensions: %lu x %lu != %lu\n", W23.rows, W23.cols, params.W23_len);
//	}

	d_mul_add(params.W23, error, params.output2);
}

__device__ void d_back_propagate_hidden(GPUTrainingParameters const params) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_back_propagate_hidden\n");
	}

	// e3 == output, see "back propagation output"
	// The weight updates are computed by
	// W23^T * e3 * ∇σ * input^T

	Matrix W23 = params.W23;
//	W23.rows = NUM_DIGITS;
//	W23.cols = params.numHiddenNodes;
//	W23.data = params.W23;
//	if (W23.rows * W23.cols != params.W23_len) {
//		printf("d_back_propagate_output: W23 matrix has wrong dimensions: %lu x %lu != %lu\n", W23.rows, W23.cols, params.W23_len);
//	}

	Matrix error = params.output3;
//	error.rows = NUM_DIGITS;
//	error.cols = params.batchSize;
//	error.data = params.output3;
//	if (error.rows * error.cols != params.output3_len) {
//		printf("d_back_propagate_output: error matrix has wrong dimensions: %lu x %lu != %lu\n", error.rows, error.cols, params.output3_len);
//	}

//	Matrix output;
//	output.rows = params.numHiddenNodes;
//	output.cols = params.batchSize;
//	output.data = params.output2;
//	if (output.rows * output.cols != params.output2_len) {
//		printf("d_back_propagate_output: output vector has wrong dimensions: %lu x %lu != %lu\n", output.rows, output.cols, params.output2_len);
//	}

//	Matrix tmp;
//	tmp.rows = params.numHiddenNodes;
//	tmp.cols = params.batchSize;
//	tmp.data = params.tmp2;
//	if (tmp.rows * tmp.cols != params.tmp2_len) {
//		printf("d_back_propagate_output: tmp vector has wrong dimensions: %lu x %lu != %lu\n", tmp.rows, tmp.cols, params.tmp2_len);
//	}

//	Matrix W12;
//	W12.rows = params.width * params.height;
//	W12.cols = params.numHiddenNodes;
//	W12.data = params.W12;
//	if (W12.rows * W12.cols != params.W12_len) {
//		printf("d_back_propagate_output: W12 has wrong dimensions: %lu x %lu != %lu\n", W12.rows, W12.cols, params.W12_len);
//	}

	Matrix images;
	images.rows = params.width * params.height;
	images.cols = params.batchSize;
	images.layout = Matrix::COLUMN_MAJOR;
	images.data = params.images;

	d_apply_activation_derivative(params.output2, params.activationFunction2);
	d_mul(params.tmp2, W23, error);
	d_cwise_mul(params.tmp2, params.output2, params.tmp2);
	d_mul_add(params.W12, params.tmp2, images);
}

__device__ void d_apply_activation(Matrix A, NeuralNetwork::ActFctType functionType) {

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_activate_layer\n");
	}

	// Target index for this thread.
	size_t const idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y + blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	// If the target index would handle an element outside of the data buffer, terminate.
	if (idx >= A.cols * A.rows) {
		return;
	}

	switch (functionType) {
	case NeuralNetwork::SIGMOID:
		A.data[idx] = 1.0f / (1.0f + exp(-A.data[idx]));
		break;
	case NeuralNetwork::TANH:
		A.data[idx] = tanh(A.data[idx]);
		break;
	}
}

__device__ void d_apply_activation_derivative(Matrix A, NeuralNetwork::ActFctType functionType) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_apply_activation_derivative\n");
	}

	// Target index for this thread.
	size_t const idx = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y + blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	// If the target index would handle an element outside of the data buffer, terminate.
	if (idx >= A.rows * A.cols) {
		return;
	}

	switch (functionType) {
	case NeuralNetwork::SIGMOID:
		A.data[idx] = A.data[idx] * (1.0f - A.data[idx]);
		break;
	case NeuralNetwork::TANH:
		float t = tanh(A.data[idx]);
		A.data[idx] = 1.0f - t * t;
		break;
	}
	//printf("actFctDeriv(%lu) = %f\n", idx, data[idx]);
}

__device__ void d_fill_target_output(GPUTrainingParameters const params, Matrix targetOutput) {

	if (targetOutput.rows != NUM_DIGITS) {
		printf("d_fill_target_output: wrong number of rows. Given %lu, expected %u\n", targetOutput.rows, NUM_DIGITS);
		return;
	}

	size_t srcIdx = threadIdx.x + blockIdx.x * blockDim.x;
	size_t targetX = threadIdx.x + blockIdx.x * blockDim.x;
	size_t targetY = threadIdx.y + blockIdx.y * blockDim.y;

	if (targetX >= targetOutput.cols || targetY >= targetOutput.rows) {
		return;
	}

//	size_t targetIdx = 0;
//	if (targetOutput.layout == Matrix::ROW_MAJOR) {
//		targetIdx = targetX + targetY * targetOutput.cols;
//	} else if (targetOutput.layout == Matrix::COLUMN_MAJOR) {
//		targetIdx = targetX * targetOutput.rows + targetY;
//	}

	//targetOutput.data[targetIdx] = (threadIdx.y == params.labels[srcIdx]) ? 1.0f : 0.0f;
	float const v = (threadIdx.y == params.labels[srcIdx]) ? 1.0f : 0.0f;
	d_matrix_set(targetOutput, targetY, targetX, v);
//	if (threadIdx.x == 0) {
//		printf("d_fill_target_output: (%lu, %lu) = %f\n", targetX, targetY, targetOutput.data[targetIdx]);
//	}
}

__device__ void d_set_bias(Matrix output, Matrix const bias) {

	if (bias.rows != output.rows) {
		printf("d_set_bias: Bias and output dimensions mismatch. Expected same height but bias was %lu and output was %lu\n", bias.rows, output.rows);
		return;
	}

	if (bias.cols > 1) {
		printf("d_set_bias: Bias column dimension is %lu > 1. Not handled.\n", bias.cols);
		return;
	}

	size_t const targetX = threadIdx.x + blockIdx.x * blockDim.x;
	size_t const targetY = threadIdx.y + blockIdx.y * blockDim.y;

	if (targetX >= output.cols || targetY >= output.rows) {
		return;
	}

	//d_matrix_set(output, targetY, targetX, d_matrix_get(bias, targetY, 1));
	d_matrix_set(output, targetY, targetX, static_cast<float>(targetY));
}

__device__ void d_assign(float* c, float const a, float const b) {
	*c = b;
}

__device__ void d_add(float* c, float const a, float const b) {
	*c = a + b;
	//printf("d_add(%f, %f, %f\n)", *a, b, c);
}

__device__ void d_sub(float* c, float const a, float const b) {
	*c = a - b;
	//printf("d_add(%f, %f, %f)\n", *c, a, b);
}

__device__ void d_mul(float* c, float const a, float const b) {
	*c = a * b;
}

__device__ void d_mul(Matrix C, Matrix const A, Matrix const B) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_mul\n");
	}
	d_mul_base(C, A, B, &d_assign);
}

__device__ void d_mul_add(Matrix C, Matrix const A, Matrix const B) {
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		printf("d_mul_add\n");
	}
	d_mul_base(C, A, B, &d_add);
}

/**
 * Computes C = AB where the dimensions of A and be have to be a multiple of MATRIX_SIZE_DIVISOR.
 *
 * @param[in] A first factor of the matrix multiplication.
 * @param[in] B second factor of the multiplication.
 * @param[out] C Matrix holding the result. Must provide enough storage space.
 */
__device__ void d_mul_base(Matrix C, Matrix const A, Matrix const B, void(*op)(float*, float const, float const)) {

	if (A.cols != B.rows) {

		printf("d_mul_base: Incompatible matrices: (%lu, %lu) x (%lu, %lu)\n", A.rows, A.cols, B.rows, B.cols);
		return;
	}

	// The block caches are row major.
	__shared__ float blockCacheA[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];
	__shared__ float blockCacheB[MATRIX_SIZE_DIVISOR][MATRIX_SIZE_DIVISOR];

	// If this thread has nothing to do, because it would access invalid memory, exit
	if (blockIdx.x * MATRIX_SIZE_DIVISOR + threadIdx.x > C.cols ||
		blockIdx.y * MATRIX_SIZE_DIVISOR + threadIdx.y > C.rows) {
		return;
	}

	float threadValue = 0.0f;
	unsigned int const numSubBlocks = A.cols / MATRIX_SIZE_DIVISOR;
	for (int k = 0; k < numSubBlocks; ++k)
	{
		if (A.layout == Matrix::COLUMN_MAJOR) {
			blockCacheA[threadIdx.y][threadIdx.x] = A.data[(blockIdx.y + k * A.cols) * MATRIX_SIZE_DIVISOR + threadIdx.y + threadIdx.x * A.cols];
		} else if (A.layout == Matrix::ROW_MAJOR) {
			blockCacheA[threadIdx.y][threadIdx.x] = A.data[(blockIdx.y * A.cols + k) * MATRIX_SIZE_DIVISOR + threadIdx.y * A.cols + threadIdx.x];
		}

		if (B.layout == Matrix::COLUMN_MAJOR) {
			blockCacheB[threadIdx.y][threadIdx.x] = B.data[(blockIdx.x * B.cols + k) * MATRIX_SIZE_DIVISOR + threadIdx.y + threadIdx.x * B.cols];
		} else if (B.layout == Matrix::ROW_MAJOR) {
			blockCacheB[threadIdx.y][threadIdx.x] = B.data[(blockIdx.x + k * B.cols) * MATRIX_SIZE_DIVISOR + threadIdx.y * B.cols + threadIdx.x];
		}

		__syncthreads();

		#pragma unroll
		for (int i = 0; i < MATRIX_SIZE_DIVISOR; ++i)
		{
			threadValue += blockCacheA[threadIdx.y][i] * blockCacheB[i][threadIdx.x];
		}

		__syncthreads();
	}

	size_t idx = 0;
	if (C.layout == Matrix::ROW_MAJOR) {
		idx = (blockIdx.y * C.cols + blockIdx.x) * MATRIX_SIZE_DIVISOR + threadIdx.y * C.cols + threadIdx.x;
	} else if (C.layout == Matrix::COLUMN_MAJOR) {
		idx = (blockIdx.y + blockIdx.x * C.cols) * MATRIX_SIZE_DIVISOR + threadIdx.y + threadIdx.x * C.cols;
	}
	float* pValue = &(C.data[idx]);
	op(pValue, *pValue, threadValue);
}

__device__ void d_cwise_sub(Matrix C, Matrix const A, Matrix const B) {
	d_cwise_op(C, A, B, &d_sub);
}

__device__ void d_cwise_mul(Matrix C, Matrix const A, Matrix const B) {
	d_cwise_op(C, A, B, &d_mul);
}

__device__ void d_cwise_op(Matrix C, Matrix const A, Matrix const B, void(*op)(float*, float const, float const)) {

	if (A.cols != B.cols || A.rows != B.rows || B.cols != C.cols || B.rows != C.rows) {

		printf("d_cwise_op: Incompatible matrices: (%lu, %lu) + (%lu, %lu) = (%lu, %lu)\n", A.rows, A.cols, B.rows, B.cols, C.rows, C.cols);
		return;
	}

	size_t const x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t const y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= A.cols || y >= A.rows) {
		return;
	}

	//C.data[idxC] = A.data[idxA] - B.data[idxB];
	op(d_matrix_pget(C, y, x), d_matrix_get(A, y, x), d_matrix_get(B, y, x));
}

__device__ void d_fill_random(Matrix A) {

	size_t const targetX = threadIdx.x + blockIdx.x * blockDim.x;
	size_t const targetY = threadIdx.y + blockIdx.y * blockDim.y;

	if (targetX >= A.cols || targetY >= A.rows) {
		return;
	}

	d_matrix_set(A, targetY, targetX, static_cast<float>(targetX));
}
