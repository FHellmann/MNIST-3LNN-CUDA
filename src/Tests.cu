#include "hip/hip_runtime.h"
/*
 * Tests.cpp
 *
 *  Created on: 14.07.2017
 *      Author: Stefan
 */
#include "NeuralNetworkParallel.h"
#include <iostream>
#include "cudaUtility.h"
#include <eigen3/Eigen/Eigen>

using namespace std;

bool ensureDeepCopy(NeuralNetworkParallel const& A, NeuralNetworkParallel const& B)
{
	if (A.learningRate != B.learningRate)
		return false;

	if (A.layers.size() != B.layers.size())
		return false;

	for (size_t i = 0; i < A.layers.size(); ++i)
	{
		NeuralNetworkParallel::Layer* layerA = A.layers[i];
		NeuralNetworkParallel::Layer* layerB = B.layers[i];
		if (layerA == layerB)
			return false;

		if (layerA->actFctType != layerB->actFctType)
			return false;
		if (layerA->layerType != layerB->layerType)
			return false;
		if (layerA->previousLayer == layerB->previousLayer)
			return false;

		for (size_t j = 0; j < A.layers.size(); ++j)
		{
			if (A.layers[j] == layerA->previousLayer)
			{
				if (B.layers[j] != layerB->previousLayer)
					return false;
			}
		}

		if (layerA->nodes.size() != layerB->nodes.size())
			return false;

		for (size_t j = 0; j < layerA->nodes.size(); ++j)
		{
			NeuralNetworkParallel::Layer::Node* nodeA = layerA->nodes[j];
			NeuralNetworkParallel::Layer::Node* nodeB = layerB->nodes[j];

			if (nodeA == nodeB)
				return false;
			if (nodeA->bias != nodeB->bias)
				return false;
			if (nodeA->output != nodeB->output)
				return false;
			if (nodeA->weights.size() != nodeB->weights.size())
				return false;

			for (size_t k = 0; k < nodeA->weights.size(); ++k)
			{
				if (nodeA->weights[k] != nodeB->weights[k])
					return false;
			}
		}
	}

	return true;
}

bool testCUDAMatrixMul() {

	Matrix d_A;
	d_A.rows = 3;
	d_A.cols = 28;
	hipMalloc((void**)&d_A.data, matrix_size(d_A) * sizeof(float));

	Matrix d_B;
	d_B.rows = d_A.cols;
	d_B.cols = 60;
	hipMalloc((void**)&d_B.data, matrix_size(d_B) * sizeof(float));

	Matrix d_C;
	d_C.rows = d_A.rows;
	d_C.cols = d_B.cols;
	hipMalloc((void**)&d_C.data, matrix_size(d_C) * sizeof(float));

	size_t largestMatDim = max(d_A.rows, d_A.cols);
	largestMatDim = max(largestMatDim, d_B.cols);
	dim3 blocks((largestMatDim - 1) / MATRIX_SIZE_DIVISOR + 1, (largestMatDim - 1) / MATRIX_SIZE_DIVISOR + 1);
	dim3 threads(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);

	printf("blocks(%u, %u), threads(%u, %u)\n", blocks.x, blocks.y, threads.x, threads.y);

	fill_pattern<<<blocks, threads>>>(d_A);
	fill_pattern<<<blocks, threads>>>(d_B);
	mul<<<blocks, threads>>>(d_C, d_A, d_B);

	typedef Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajorBit> EigenMatrix;
	EigenMatrix A(d_A.rows, d_A.cols);
	EigenMatrix B(d_B.rows, d_B.cols);
	EigenMatrix C(d_C.rows, d_C.cols);

	hipMemcpy((void**)A.data(), d_A.data, matrix_size(d_A) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy((void**)B.data(), d_B.data, matrix_size(d_B) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy((void**)C.data(), d_C.data, matrix_size(d_C) * sizeof(float), hipMemcpyDeviceToHost);

	printf("A(%lu, %lu)\n", A.rows(), A.cols());
	cout << A << endl << endl;
	printf("B(%lu, %lu)\n", B.rows(), B.cols());
	cout << B << endl << endl;
	printf("C(%lu, %lu)\n", C.rows(), C.cols());
	cout << C << endl << endl;

	hipFree(d_A.data);
	hipFree(d_B.data);
	hipFree(d_C.data);

	return (A*B).isApprox(C);
}

bool testCUDAMatrixMulTranspose() {

	Matrix d_A;
	d_A.rows = 10;
	d_A.cols = 60;
	hipMalloc((void**)&d_A.data, matrix_size(d_A) * sizeof(float));

	Matrix d_B;
	d_B.rows = 28;
	d_B.cols = d_A.cols;
	hipMalloc((void**)&d_B.data, matrix_size(d_B) * sizeof(float));


	size_t largestMatDim = 0;
	largestMatDim = max(largestMatDim, d_A.rows);
	largestMatDim = max(largestMatDim, d_A.cols);
	largestMatDim = max(largestMatDim, d_B.rows);
	largestMatDim = max(largestMatDim, d_B.cols);
	dim3 blocks((largestMatDim - 1) / MATRIX_SIZE_DIVISOR + 1, (largestMatDim - 1) / MATRIX_SIZE_DIVISOR + 1);
	dim3 threads(MATRIX_SIZE_DIVISOR, MATRIX_SIZE_DIVISOR);

	printf("blocks(%u, %u), threads(%u, %u)\n", blocks.x, blocks.y, threads.x, threads.y);

	fill_pattern<<<blocks, threads>>>(d_A);
	fill_pattern<<<blocks, threads>>>(d_B);
	d_B = matrix_transpose(d_B);

	Matrix d_C;
	d_C.rows = d_A.rows;
	d_C.cols = d_B.cols;
	hipMalloc((void**)&d_C.data, matrix_size(d_C) * sizeof(float));
	fill<<<blocks, threads>>>(d_C, 0.0f);
	mul_add<<<blocks, threads>>>(d_C, d_A, d_B);

	typedef Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> EigenMatrixRowMajor;
	typedef Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::ColMajor> EigenMatrixColumnMajor;
	EigenMatrixRowMajor A(d_A.rows, d_A.cols);
	EigenMatrixColumnMajor B(d_B.rows, d_B.cols);
	//EigenMatrixRowMajor B(d_B.rows, d_B.cols);
	EigenMatrixRowMajor C(d_C.rows, d_C.cols);

	hipMemcpy((void**)A.data(), d_A.data, matrix_size(d_A) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy((void**)B.data(), d_B.data, matrix_size(d_B) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy((void**)C.data(), d_C.data, matrix_size(d_C) * sizeof(float), hipMemcpyDeviceToHost);

	printf("A(%lu, %lu)\n", A.rows(), A.cols());
	cout << A << endl << endl;
	printf("B(%lu, %lu)\n", B.rows(), B.cols());
	cout << B << endl << endl;
	printf("C(%lu, %lu)\n", C.rows(), C.cols());
	cout << C << endl << endl;

	hipFree(d_A.data);
	hipFree(d_B.data);
	hipFree(d_C.data);

	return (A*B).isApprox(C);
}


int main(int argc, char* argv[])
{
	NeuralNetworkParallel A(4, 2, 17, 0.2);
	NeuralNetworkParallel B(A);

	if (ensureDeepCopy(A, B) == false)
	{
		cerr << "B is not a deep copy of A!" << endl;
		exit (EXIT_FAILURE);
	}

	if (!testCUDAMatrixMulTranspose()) {
		cerr << "Matrix multiplication errornous." << endl;
		exit (EXIT_FAILURE);
	}

	exit (EXIT_SUCCESS);
}
